#include "hip/hip_runtime.h"
// Copyright (C) 2012 von Karman Institute for Fluid Dynamics, Belgium
//
// This software is distributed under the terms of the
// GNU Lesser General Public License version 3 (LGPLv3).
// See doc/lgpl.txt and doc/gpl.txt for the license text.

#include "Framework/CudaDeviceManager.hh"
#include "Common/CFLog.hh"
#include "Common/PE.hh"
	
//////////////////////////////////////////////////////////////////////////////

namespace COOLFluiD {

  namespace CudaEnv {
          
//////////////////////////////////////////////////////////////////////////////

void CudaDeviceManager::defineConfigOptions(Config::OptionList& options)
{
  options.addConfigOption< CFuint >("NbThreadsPerBlock", "Number if threads per block");
  options.addConfigOption< CFuint >("NbBlocks", "Number of blocks");
}
    
//////////////////////////////////////////////////////////////////////////////

CudaDeviceManager::CudaDeviceManager() : Config::ConfigObject("CudaDeviceManager"), 
					 Common::NonCopyable<CudaDeviceManager>()
{
  addConfigOptionsTo(this);

  NTHREADS_PER_BLOCK = 0;
  setParameter("NbThreadsPerBlock", &NTHREADS_PER_BLOCK);
  
  NBLOCKS = 0;
  setParameter("NbBlocks", &NBLOCKS);
}
    
//////////////////////////////////////////////////////////////////////////////
 
void CudaDeviceManager::configure ( Config::ConfigArgs& args )
{
  CFLog(VERBOSE, "CudaDeviceManager::configure() START\n");
  
  ConfigObject::configure(args);
  
  CFLog(VERBOSE, "CudaDeviceManager::configure() => hipGetDeviceCount()\n");
  
  // the first time getInstance() is called, the device is initialized
  // for the moment we assume one device
  int count = 0; 
  int dev   = -1;
  int rank = Common::PE::GetPE().GetRank("Default");
  hipGetDeviceCount(&count);
 
  const CFuint nbProc = Common::PE::GetPE().GetProcessorCount("Default"); 
  if (nbProc > 1) {
    dev = rank % count;
    cfassert(dev >= 0); 
    hipSetDevice(dev);
    //    std::cout << "P" << rank << " => device/count => " << dev << "/" << count << std::endl;
    // std::cout << "CudaDeviceManager::configure() => infinite loop ...\n"; 
    // for (;;) {}
  } else {
     /* the code below works for serial GPU simulations */
    hipGetDevice(&dev);
    cfassert(dev >= 0);  
    hipSetDevice(dev);
  }
  
  CFLog(VERBOSE, "CudaDeviceManager::configure() => hipGetDeviceProperties()\n");
  hipGetDeviceProperties(&m_prop, dev); 
  
  CFLog(VERBOSE, "CudaDeviceManager::configure() => printProperties()\n");
  printProperties(dev);
  
  NTHREADS_PER_BLOCK = (NTHREADS_PER_BLOCK > 0) ? NTHREADS_PER_BLOCK :  m_prop.maxThreadsPerBlock;
  cfassert(NTHREADS_PER_BLOCK > 0);
  
  NBLOCKS = (NBLOCKS > 0) ? NBLOCKS : m_prop.maxGridSize[0];
  cfassert(NBLOCKS > 0);
  
  CFLog(INFO, "CudaDeviceManager::configure() => NTHREADS_PER_BLOCK = " << NTHREADS_PER_BLOCK 
	<< ", NBLOCKS = " << NBLOCKS << " END\n");
  
  CFLog(VERBOSE, "CudaDeviceManager::configure() END\n");
}
    
//////////////////////////////////////////////////////////////////////////////
 
void CudaDeviceManager::printProperties(int dev)
{
  using namespace std;
  
  CFLog(INFO, "##### CudaDeviceManager::printProperties() for device [" << dev << "] #####\n");
  CFLog(INFO, "name = " << m_prop.name << "\n");
  CFLog(INFO, "capability = " << m_prop.major << "." << m_prop.minor << "\n"); 
  CFLog(INFO, "clock rate = " << m_prop.clockRate <<"\n"); 
  CFLog(INFO, "total global mem = " << m_prop.totalGlobalMem <<"\n"); 
  CFLog(INFO, "total constant mem = " << m_prop.totalConstMem <<"\n");
  CFLog(INFO, "overlap execution and transfer = "); 
  if (m_prop.deviceOverlap) {
    CFLog(INFO, "ENABLED\n");
  }
  else {
    CFLog(INFO, "DISABLED\n");
  }
  CFLog(INFO, "can map host memory = "); 
  if (m_prop.canMapHostMemory) {
    CFLog(INFO, "ENABLED\n");
  }
  else {
    CFLog(INFO, "DISABLED\n");
  }
  CFLog(INFO, "texture alignment = " << m_prop.textureAlignment << "\n");
  CFLog(INFO, "multiprocessor count = " << m_prop.multiProcessorCount << "\n");
  CFLog(INFO, "shared mem per block = " << m_prop.sharedMemPerBlock << "\n");
  CFLog(INFO, "registers per block = " << m_prop.regsPerBlock << "\n");
  CFLog(INFO, "threads in warp = " << m_prop.warpSize << "\n");
  CFLog(INFO, "max threads per block = " << m_prop.maxThreadsPerBlock << "\n");
  CFLog(INFO, "max threads dimensions = " << m_prop.maxThreadsDim[0] << " " 
       <<  m_prop.maxThreadsDim[1] << " " <<  m_prop.maxThreadsDim[2] << "\n");
  CFLog(INFO, "max grid dimensions = " << m_prop.maxGridSize[0] << " " 
       <<  m_prop.maxGridSize[1] << " " <<  m_prop.maxGridSize[2] << "\n");
  CFLog(INFO,  "############################################################\n\n"); 
}

//////////////////////////////////////////////////////////////////////////////

} // end namespace CudaEnv 
  
} // end  namespace COOLFluiD

//////////////////////////////////////////////////////////////////////////////
