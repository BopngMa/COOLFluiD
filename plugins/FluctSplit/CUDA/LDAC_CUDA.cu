#include "hip/hip_runtime.h"
#include "FluctSplit/CUDA/LDAC_CUDA.hh"
#include "Framework/CudaDeviceManager.hh"

//////////////////////////////////////////////////////////////////////////////

namespace COOLFluiD {
  
    namespace FluctSplit {

//////////////////////////////////////////////////////////////////////////////
      
__global__ void addToKPlus(int msize, double* a,  double* b, double* c, double* d) 
{
  int tid = threadIdx.x; // + blockIdx.x * blockDim.x;
  if (tid < msize) {
    d[tid] = a[tid] + b[tid] + c[tid];
    // tid += blockDim.x*gridDim.x;`
  }
}

//////////////////////////////////////////////////////////////////////////////
      
void addToKplusCUDA(int msize, 
		    double* dev_a,  double* dev_b, double* dev_c, double* dev_d, 
		    double* k0,  double* k1,  double* k2, double* sumKplus)
{
  using namespace std;
  
  CUDA_CHECK(hipMemcpy(dev_a, k0, msize*sizeof(double), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(dev_b, k1, msize*sizeof(double), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(dev_c, k2, msize*sizeof(double), hipMemcpyHostToDevice));
  
  addToKPlus<<<1,msize>>>(msize, dev_a, dev_b, dev_c, dev_d);
  
  CUDA_CHECK(hipMemcpy(sumKplus, dev_d, msize*sizeof(double), hipMemcpyDeviceToHost));
}

//////////////////////////////////////////////////////////////////////////////

    } // namespace FluctSplit

} // namespace COOLFluiD

//////////////////////////////////////////////////////////////////////////////
