#include "hip/hip_runtime.h"
#include <paralution.hpp>
#include "ParalutionMatrix.hh"

#include "Framework/BlockAccumulatorBaseCUDA.hh"
#include "Framework/BlockAccumulator.hh"
#include "Framework/CudaDeviceManager.hh"
#include "Common/CUDA/CFVec.hh"
#include "FiniteVolume/CellData.hh"
#include "Common/CUDA/CudaEnv.hh"
#include "Framework/CudaDeviceManager.hh"


using namespace COOLFluiD::Framework;
//using namespace COOLFluiD::Common;
//using namespace COOLFluiD::Config;
using namespace std;

namespace COOLFluiD {

    namespace Paralution {

/*
__global__ void addDiagBlockGPU(CFreal *val, CFint *rowoff, CFint *col, CFreal *accDevPtr, CFuint *nbs, CFint *IDs)
{
  const int nbj = threadIdx.x; //+ blockIdx.x*blockDim.x;
  const int nbi = threadIdx.y; // + blockIdx.x*blockDim.x;
  //printf("nbi %d \t nbj %d \n", nbi, nbj);
  //if(cellID!=0){printf("inside addDiagBlockGPU %d \n", cellID);}
  //if (cellID < nbCells) { 
    const CFint nb = nbs[0];

    CFuint RowPositionDiag = rowoff[IDs[0]*nb];           //In this case we are looking for the diagonal block
    CFuint RowPositionPlusOneDiag = rowoff[IDs[0]*nb + 1];
    CFuint mmDiag = (RowPositionPlusOneDiag-RowPositionDiag)/nb;
    CFuint IndexCSRDiag = -1;

          IndexCSRDiag = RowPositionDiag; //+mii*nb;

    //for (CFint nbj=0; nbj<nb; nbj++){
      //for (CFint nbi=0; nbi<nb; nbi++){
        val[IndexCSRDiag+nbi*nb*mmDiag+nbj] += accDevPtr[nbi*nb + nbj];
     // }
    //}
  //}
} 
*/

__global__ void addDiagBlockGPU(CFreal *val, CFint *rowoff, CFreal *accDevPtr, CFuint nb, CFuint startCellID, CFint nbCells)
{
  const int localID = threadIdx.x + blockIdx.x*blockDim.x;
  const int cellID = localID + startCellID;
  if (cellID < nbCells) { 
    CFuint RowPositionDiag = rowoff[cellID*nb];
    CFuint RowPositionPlusOneDiag = rowoff[cellID*nb + 1];
    CFuint mmDiag = (RowPositionPlusOneDiag-RowPositionDiag)/nb;
    for(CFint nbi=0; nbi<nb; nbi++){
      for(CFint nbj=0; nbj<nb; nbj++){
          val[RowPositionDiag+nbi*nb*mmDiag+nbj] += accDevPtr[localID*nb*nb + nbi*nb + nbj];
      }
    }
  }
}

////////////////////////////////////////////////////////////////////

void ParalutionMatrix::resetToZeroEntriesGPU(){
   //std::cout << "resetToZeroEntriesGPU \n";
   //CFuint nbThreadPerBlock = 64;
   //CFuint nbBlocks = _size/64;
hipMemset(_valDev, 0.0, _size*sizeof(CFreal));
std::fill_n(_diagAcc, diagAccSize, 0);
   //ZeroEntriesGPU<<< nbBlocks,nbThreadPerBlock >>> (_valDev, _size);
}

////////////////////////////////////////////////////////////////////

void ParalutionMatrix::addValuesGPU(const Framework::BlockAccumulator& acc)
{
   CFuint nb = acc.getNB();
   CFreal* accPtr = const_cast<Framework::BlockAccumulator&>(acc).getPtr();
   CFint IDs = const_cast<std::vector<CFint>&>(acc.getIN())[0]; //Array storing cellID

   CFuint cellIndex = IDs*nb*nb;
   for(CFint nbi=0; nbi<nb; nbi++){
     for(CFint nbj=0; nbj<nb; nbj++){
        _diagAcc[cellIndex + nbi*nb + nbj] += accPtr[nbi*nb + nbj];
     }
   } 
}  


void ParalutionMatrix::updateDiagBlocks(CFuint nbCells, CFuint nbEqs)
{
   //Copy the array to GPU

   CFuint startCellID = 0;
   for (CFuint s = 0; s < _sizeb; ++s) {
      CudaEnv::copyHost2Dev(_diagAccDev, &_diagAcc[startCellID*nbEqs*nbEqs], _nThreads*_nbKernelBlocks*nbEqs*nbEqs);
      addDiagBlockGPU <<<_nbKernelBlocks,_nThreads>>> (_valDev, _rowoffDev, _diagAccDev, nbEqs, startCellID, nbCells);
 
      startCellID += _nThreads*_nbKernelBlocks; //m_nbCellsInKernel[s];
   }

}

/*
void ParalutionMatrix::addValuesGPU(const Framework::BlockAccumulator& acc)
{
   //const CFuint blocksPerGrid = CudaEnv::CudaDeviceManager::getInstance().getBlocksPerGrid(nbCells);
   //const CFuint nThreads = CudaEnv::CudaDeviceManager::getInstance().getNThreads();

   //Always m=n=1!! -> More efficient algorithm

   //Read the data from the BlockAccumulator
    CFuint size = acc.size();
   // CFuint m = acc.getM();
   // CFuint n = acc.getN();
    CFuint nb = acc.getNB();
   CFint nbID = const_cast<std::vector<CFint>&>(acc.getIM())[0]; //Array storing indexes of neigbours
   CFint IDs = const_cast<std::vector<CFint>&>(acc.getIN())[0]; //Array storing cellID
   CFreal* accPtr = const_cast<Framework::BlockAccumulator&>(acc).getPtr();

   //Create the device pointers
   CFreal* accDevPtr;
   //CFint* nbIDDev;
   CFint* IDsDev;
   CFuint* nbDev;

   //Allocate
   CudaEnv::allocDev(accDevPtr, size);
   CudaEnv::allocDev(nbDev, 1);
   CudaEnv::allocDev(IDsDev, 1);

   //Copy the data
   CudaEnv::copyHost2Dev(accDevPtr, accPtr, size);
   CudaEnv::copyHost2Dev(nbDev, &nb, 1);
   CudaEnv::copyHost2Dev(IDsDev, &IDs, 1);
   //CudaEnv::copyHost2Dev(mDev, &m, 1);
   //CudaEnv::copyHost2Dev(nDev, &n, 1);

   //Call kernel to add values
   //if(IDs != nbID){printf("IDs %d \t nbIDs %d \n", IDs, nbID);}
   dim3 threads(nb, nb);
   addDiagBlockGPU<<<1,threads>>> (_valDev, _rowoffDev, _colDev, accDevPtr, nbDev, IDsDev);

   // Deallocate memory GPU
   CudaEnv::free(accDevPtr);
   CudaEnv::free(nbDev);
   CudaEnv::free(IDsDev);
   //CudaEnv::free(mDev);
   //CudaEnv::free(nDev);
}
*/


    }   // namespace COOLFluiD

}  // namespace Paralution
