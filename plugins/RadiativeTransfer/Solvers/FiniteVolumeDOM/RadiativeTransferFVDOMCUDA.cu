#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>

#include "Common/PE.hh"
#include "Common/BadValueException.hh"
#include "Common/CFPrintContainer.hh"
#include "Framework/CudaDeviceManager.hh"
#include "Common/CUDA/CFVec.hh"
#include "Framework/CudaTimer.hh"

#include "MathTools/MathConsts.hh"

#include "Environment/ObjectProvider.hh"
#include "Environment/CFEnv.hh"
#include "Environment/FileHandlerOutput.hh"
#include "Environment/FileHandlerInput.hh"
#include "Environment/DirPaths.hh"
#include "Environment/SingleBehaviorFactory.hh"

#include "Framework/PathAppender.hh"
#include "Framework/DataProcessing.hh"
#include "Framework/SubSystemStatus.hh"
#include "Framework/MethodCommandProvider.hh"
#include "Framework/MeshData.hh"
#include "Framework/PhysicalChemicalLibrary.hh"
#include "Framework/SocketBundleSetter.hh"

#include "FiniteVolume/CellCenterFVM.hh"

#include "RadiativeTransfer/RadiativeTransfer.hh"
#include "RadiativeTransfer/Solvers/FiniteVolumeDOM/RadiativeTransferFVDOMCUDA.hh"
#include "RadiativeTransfer/RadiationLibrary/Radiator.hh"
#include "RadiativeTransfer/RadiationLibrary/RadiationPhysicsHandler.hh"

/////////////////////////////////////////////////////////////////////////////

using namespace std;
using namespace COOLFluiD::Framework;
using namespace COOLFluiD::Common;
using namespace COOLFluiD::MathTools;
using namespace COOLFluiD::Numerics::FiniteVolume;

//////////////////////////////////////////////////////////////////////////////

namespace COOLFluiD {

    namespace RadiativeTransfer {

//////////////////////////////////////////////////////////////////////////////

MethodCommandProvider<RadiativeTransferFVDOMCUDA, 
		      DataProcessingData, 
		      RadiativeTransferModule>
radiativeTransferFVDOMCUDAProvider("RadiativeTransferFVDOMCUDA");

//////////////////////////////////////////////////////////////////////////////

__global__ void getFieldOpacitiesKernel(const bool useExponentialMethod,
					const CFuint TID, 
					const CFuint PID,
					const CFuint nbTemp,
					const CFuint nbPress,
					const CFuint nbBins,
					const CFuint ib,
					const CFuint nbEqs,
					const CFuint nbCells,
					const CFreal* states,
					const CFreal* volumes,
					const CFreal* Ttable,
					const CFreal* Ptable,
					const CFreal* opacities,
					const CFreal* radSource,
					CFreal* fieldSource,
					CFreal* fieldAbsor,
					CFreal* fieldAbSrcV,
					CFreal* fieldAbV)
{    
  // each thread takes care of computing the gradient for one single cell
  const int cellID = threadIdx.x + blockIdx.x*blockDim.x;
  
  if (cellID < nbCells) {
    fieldSource[cellID] = 0.;
    if(useExponentialMethod) {
      fieldAbsor[cellID]  = 0.;
    }
    else {
      fieldAbSrcV[cellID] = 0.;
      fieldAbV[cellID]    = 0.;
    }
    
    //Get the field pressure and T commented because now we impose a temperature profile
    const CFuint sIdx = cellID*nbEqs; 
    const CFreal p = states[sIdx + PID];
    const CFreal T = states[sIdx + TID];
    const CFreal patm = p/101325.; //converting from Pa to atm
    
    CFreal val1 = 0;
    CFreal val2 = 0;
    
    RadiativeTransferFVDOM::Interpolator interp;
    interp.tableInterpolate(nbBins, nbTemp, nbPress, Ttable, Ptable,
			    opacities, radSource, T, patm, ib, val1, val2); 
    
    if(useExponentialMethod){
      if (val1 <= 1e-30 || val2 <= 1e-30 ){
	fieldSource[cellID] = 1e-30;
	fieldAbsor[cellID]  = 1e-30;
      }
      else {
	fieldSource[cellID] = val2/val1;
	fieldAbsor[cellID]  = val1;
      }
    }
    else{
      if (val1 <= 1e-30 || val2 <= 1e-30 ){
	fieldSource[cellID] = 1e-30;
	fieldAbV[cellID]    = 1e-30*volumes[cellID]; // Volumen converted from m^3 into cm^3
      }
      else {
	fieldSource[cellID] = val2/val1;
	fieldAbV[cellID]    = val1*volumes[cellID];
      }      
      fieldAbSrcV[cellID]   = fieldSource[cellID]*fieldAbV[cellID];
    }
  }
}

//////////////////////////////////////////////////////////////////////////////

__global__ void computeQKernelExponential(const CFuint d,
					  const CFuint nbCells,
					  const CFreal weightIn,
					  const CFuint* cellFaces,
					  const CFint* faceCell,
					  const CFuint* nbFacesInCell,
					  const CFint* isOutward,
					  const CFint* advanceOrder,
					  const CFreal* volumes,
					  const CFreal* fieldSource,
					  const CFreal* fieldAbsor,
					  const CFreal* dotProdInFace,
					  const CFreal* mdirs,
					  CFreal* In,
					  CFreal* II,
					  CFreal* divQ,
					  CFreal* qx, CFreal* qy, CFreal* qz)
{    
  // each thread takes care of computing the gradient for one single cell
  const int cellID = threadIdx.x + blockIdx.x*blockDim.x;
  
  __shared__ CFreal weight;
  weight = weightIn;
  
  if (cellID < nbCells) {
    // allocate the cell entity
    const CFuint iCell   = abs(advanceOrder[cellID]);
    CFreal Ic            = 0.;
    CFreal inDirDotnANeg = 0.;
    CFreal dirDotnANeg   = 0;
    CFreal Lc            = 0;
    CFreal halfExp       = 0;
    
    const CFuint nbFaces = nbFacesInCell[iCell];
    for (CFuint iFace = 0; iFace < nbFaces; ++iFace) { 
      const CFuint faceID = cellFaces[iFace*nbCells + iCell];
      const CFreal factor = ((CFuint)(isOutward[faceID]) != iCell) ? -1. : 1.;
      const CFreal dirDotNA = dotProdInFace[faceID]*factor;
      
      if(dirDotNA < 0.) {
	dirDotnANeg += dirDotNA;
	
        const CFint fcellID = faceCell[faceID*2]; 
        const CFint neighborCellID = (fcellID == iCell) ? faceCell[faceID*2+1] : fcellID;
	const CFreal source = (neighborCellID >=0) ? In[neighborCellID] : fieldSource[iCell];
        inDirDotnANeg += source*dirDotNA;
	
	/*const bool isBFace = mapGeoToTrs->isBGeo(faceID); /////
	  if (!isBFace){
	  const CFuint neighborCellID = getNeighborCellID(faceID, iCell); /////
	  inDirDotnANeg += In[neighborCellID]*dirDotNA;
	  }
	  else {
	  const CFreal boundarySource = fieldSource[iCell];
	  inDirDotnANeg += boundarySource*dirDotNA;
	  }*/
      }
    } 
    Lc        = volumes[iCell]/(- dirDotnANeg); 
    halfExp   = std::exp(-0.5*Lc*fieldAbsor[iCell]);
    In[iCell] = (inDirDotnANeg/dirDotnANeg)*halfExp*halfExp + (1. - halfExp*halfExp)*fieldSource[iCell];
    Ic        = (inDirDotnANeg/dirDotnANeg)*halfExp + (1. - halfExp)*fieldSource[iCell];
    
    qx[iCell] += Ic*mdirs[d*3]*weight;
    qy[iCell] += Ic*mdirs[d*3+1]*weight;
    qz[iCell] += Ic*mdirs[d*3+2]*weight;
    
    CFreal inDirDotnA = inDirDotnANeg;
    for (CFuint iFace = 0; iFace < nbFaces; ++iFace) {
      const CFuint faceID = cellFaces[iFace*nbCells + iCell];
      const CFreal factor = ((CFuint)(isOutward[faceID]) != iCell) ? -1. : 1.;
      const CFreal dirDotNA = dotProdInFace[faceID]*factor;
      if (dirDotNA > 0.) {
	inDirDotnA += In[iCell]*dirDotNA;
      }
    }
    
    divQ[iCell] += inDirDotnA*weight;
    II[iCell]   += Ic*weight;
  }  
}
      
//////////////////////////////////////////////////////////////////////////////

__global__ void computeQKernelNoExponential(const CFuint d, 
					    const CFuint nbCells,
					    const CFreal weightIn,
					    const CFuint* cellFaces,
					    const CFint* faceCell,
					    const CFuint* nbFacesInCell,
					    const CFint* isOutward,
					    const CFint* advanceOrder,
					    const CFreal* volumes,
					    const CFreal* fieldSource,
					    const CFreal* fieldAbSrcV,
					    const CFreal* fieldAbV,
					    const CFreal* dotProdInFace,
					    const CFreal* mdirs,
					    CFreal* In,
					    CFreal* II,
					    CFreal* divQ,
					    CFreal* qx, CFreal* qy, CFreal* qz)
{    
  // each thread takes care of computing the gradient for one single cell
  const int cellID = threadIdx.x + blockIdx.x*blockDim.x;
  
  __shared__ CFreal weight;
  weight = weightIn;
  
  if (cellID < nbCells) {
    // allocate the cell entity
    const CFuint iCell = abs(advanceOrder[cellID]);
    CFreal inDirDotnANeg = 0.;
    CFreal Ic            = 0.;
    CFreal dirDotnAPos   = 0;
    
    const CFuint nbFaces = nbFacesInCell[iCell];
    for (CFuint iFace = 0; iFace < nbFaces; ++iFace) {
      const CFuint faceID = cellFaces[iFace*nbCells + iCell];
      const CFreal factor = ((CFuint)(isOutward[faceID]) != iCell) ? -1. : 1.;
      const CFreal dirDotNA = dotProdInFace[faceID]*factor;
      
      if (dirDotNA >= 0.){
	dirDotnAPos += dirDotNA;
      }
      else {
	const CFint fcellID = faceCell[faceID*2]; 
        const CFint neighborCellID = (fcellID == iCell) ? faceCell[faceID*2+1] : fcellID;
	const CFreal source = (neighborCellID >=0) ? In[neighborCellID] : fieldSource[iCell];
        inDirDotnANeg += source*dirDotNA;
      }
    } 
    In[iCell] = (fieldAbSrcV[iCell] - inDirDotnANeg)/(fieldAbV[iCell] + dirDotnAPos);
    Ic = In[iCell];
    
    qx[iCell] += Ic*mdirs[d*3]*weight;
    qy[iCell] += Ic*mdirs[d*3+1]*weight;
    qz[iCell] += Ic*mdirs[d*3+2]*weight;
    
    CFreal inDirDotnA = inDirDotnANeg;
    for (CFuint iFace = 0; iFace < nbFaces; ++iFace) {
      const CFuint faceID = cellFaces[iFace*nbCells + iCell];
      const CFreal factor = ((CFuint)(isOutward[faceID]) != iCell) ? -1. : 1.;
      const CFreal dirDotNA = dotProdInFace[faceID]*factor;
      if (dirDotNA > 0.) {
	inDirDotnA += In[iCell]*dirDotNA;
      }
    }
    
    divQ[iCell] += inDirDotnA*weight;
    II[iCell] += Ic*weight;
  }  
}
      
//////////////////////////////////////////////////////////////////////////////

RadiativeTransferFVDOMCUDA::RadiativeTransferFVDOMCUDA(const std::string& name) :
  RadiativeTransferFVDOM(name),
  m_faceCell(),
  m_nbFacesInCell()
{
  addConfigOptionsTo(this);
}
      
//////////////////////////////////////////////////////////////////////////////

RadiativeTransferFVDOMCUDA::~RadiativeTransferFVDOMCUDA()
{
}

//////////////////////////////////////////////////////////////////////////////

void RadiativeTransferFVDOMCUDA::defineConfigOptions(Config::OptionList& options)
{  
}
      
//////////////////////////////////////////////////////////////////////////////

void RadiativeTransferFVDOMCUDA::setup()
{
  CFAUTOTRACE;
    
  RadiativeTransferFVDOM::setup();
  
  CFLog(VERBOSE, "RadiativeTransferFVDOMCUDA::setup() => START\n");
  
  // store invariant data on GPU
  SafePtr<ConnectivityTable<CFuint> > cellFaces = 
    MeshDataStack::getActive()->getConnectivity("cellFaces");
  cellFaces->getPtr()->put();
  socket_isOutward.getDataHandle().getLocalArray()->put(); 
  socket_volumes.getDataHandle().getLocalArray()->put(); 
  m_dirs.put();
  m_fieldSource.put(); // to be removed
  m_fieldAbsor.put();  // to be removed
  m_fieldAbSrcV.put(); // to be removed
  m_fieldAbV.put();  // to be removed
  m_In.put(); // to be removed
  m_II.put(); // to be removed
  m_opacities.put();
  m_radSource.put();
  m_Ttable.put();
  m_Ptable.put();
  
  const CFuint totalNbFaces = MeshDataStack::getActive()->Statistics().getNbFaces();
  m_faceCell.resize(totalNbFaces*2);
  m_faceCell = -1;
  
  const CFuint nbCells = socket_volumes.getDataHandle().size();
  m_nbFacesInCell.resize(nbCells);
  
  for (CFuint iCell = 0; iCell < nbCells; ++iCell) {
    const CFuint nbFaces = cellFaces->nbCols(iCell);
    m_nbFacesInCell[iCell] = nbFaces;
    for (CFuint iFace = 0; iFace < nbFaces; ++iFace) {
      const CFuint faceID2 = (*cellFaces)(iCell, iFace)*2;
      if (m_faceCell[faceID2] == -1) {
	m_faceCell[faceID2] = iCell;
      }
      else {
	m_faceCell[faceID2+1] = iCell;
      }
    }
  }
  
  m_faceCell.put();
  m_nbFacesInCell.put();
  
  CFLog(VERBOSE, "RadiativeTransferFVDOMCUDA::setup() => END\n");
}
      
//////////////////////////////////////////////////////////////////////////////

void RadiativeTransferFVDOMCUDA::unsetup()
{
  CFAUTOTRACE;
  
  RadiativeTransferFVDOM::unsetup();
}
      
//////////////////////////////////////////////////////////////////////////////
 
void RadiativeTransferFVDOMCUDA::loopOverDirs(const CFuint startBin, 
					      const CFuint endBin, 
					      const CFuint startDir,
					      const CFuint endDir)
{
  CFLog(VERBOSE, "RadiativeTransferFVDOMCUDA::loopOverDirs() => START\n");
  
  SafePtr<ConnectivityTable<CFuint> > cellFaces = 
    MeshDataStack::getActive()->getConnectivity("cellFaces");
  DataHandle<CFint> isOutward = socket_isOutward.getDataHandle();
  DataHandle<CFreal> volumes = socket_volumes.getDataHandle();
  DataHandle<CFreal> divQ = socket_divq.getDataHandle();
  DataHandle<State*, GLOBAL> states = socket_states.getDataHandle();
  DataHandle<CFreal> qx = socket_qx.getDataHandle();
  DataHandle<CFreal> qy = socket_qy.getDataHandle();
  DataHandle<CFreal> qz = socket_qz.getDataHandle();
  
  const CFuint nbCells = states.size();
  const CFuint nbEqs = PhysicalModelStack::getActive()->getNbEq();
  
  CFLog(VERBOSE, "RadiativeTransferFVDOMCUDA::loopOverDirs() => 0\n");
  
  states.getGlobalArray()->put();
  
  CFLog(VERBOSE, "RadiativeTransferFVDOMCUDA::loopOverDirs() => 1\n");
  
  const CFuint blocksPerGrid = 
    CudaEnv::CudaDeviceManager::getInstance().getBlocksPerGrid(nbCells);
  const CFuint nThreads = CudaEnv::CudaDeviceManager::getInstance().getNThreads();
  
  // if more than one iteration is needed, the initialization has to be done here
  // for the moment it is done the RadiativeTransferFVDOM::setup();
  // divq = 0.; qx = 0.; qy = 0.; qz = 0.; 
  socket_divq.getDataHandle().getLocalArray()->put();
  socket_qx.getDataHandle().getLocalArray()->put();
  socket_qy.getDataHandle().getLocalArray()->put();
  socket_qz.getDataHandle().getLocalArray()->put();
  
  CFLog(VERBOSE, "RadiativeTransferFVDOMCUDA::loopOverDirs() => 2\n");
  
  for (CFuint d = startDir; d < endDir; ++d) {
    CFLog(INFO, "( dir: " << d << " ), ( bin: ");
    const CFuint bStart = (d != startDir) ? 0 : startBin;
    const CFuint bEnd   = (d != m_startEndDir.second) ? m_nbBins : endBin;
    
    // this is buggy!!!!!!!
    m_advanceOrder[d].put();
    
    CFLog(VERBOSE, "RadiativeTransferFVDOMCUDA::loopOverDirs() => 3\n");
    
    // precompute dot products for all faces and directions (a part from the sign)
    computeDotProdInFace(d, m_dotProdInFace);
    m_dotProdInFace.put();
    
    CFLog(VERBOSE, "RadiativeTransferFVDOMCUDA::loopOverDirs() => 4\n");
    
    for (CFuint ib = startBin; ib < endBin; ++ib) {
      CFLog(INFO, "[dir, bin] = [" << d << ", " << ib << "]\n");
      
      // precompute the radiation properties for all cells
      getFieldOpacitiesKernel<<<blocksPerGrid,nThreads>>>
	(m_useExponentialMethod, 
	 m_TID, m_PID, m_nbTemp, m_nbPress, m_nbBins,
	 ib, nbEqs, nbCells, m_Ttable.ptrDev(), m_Ptable.ptrDev(), 
	 states.getGlobalArray()->ptrDev(),
	 volumes.getLocalArray()->ptrDev(),
	 m_opacities.ptrDev(),
	 m_radSource.ptrDev(),
	 m_fieldSource.ptrDev(),
	 m_fieldAbsor.ptrDev(),
	 m_fieldAbSrcV.ptrDev(),
	 m_fieldAbV.ptrDev());  
      
      m_fieldSource.get();
      m_fieldAbsor.get();
      m_fieldAbSrcV.get();
      m_fieldAbV.get();
      
      RadiativeTransferFVDOM::computeQ(ib,d);
      
      // compute the radiative heat flux
      /*  if (m_useExponentialMethod) {
	computeQKernelExponential<<<blocksPerGrid,nThreads>>> 
	  (d, nbCells, m_weight[d],
	   cellFaces->getPtr()->ptrDev(),
	   m_faceCell.ptrDev(),
	   m_nbFacesInCell.ptrDev(),
	   isOutward.getLocalArray()->ptrDev(),
	   m_advanceOrder[d].ptrDev(),
	   volumes.getLocalArray()->ptrDev(),
	   m_fieldSource.ptrDev(),
	   m_fieldAbsor.ptrDev(),
	   m_dotProdInFace.ptrDev(),
	   m_dirs.ptrDev(),
	   m_In.ptrDev(), m_II.ptrDev(), 
	   divQ.getLocalArray()->ptrDev(),
	   qx.getLocalArray()->ptrDev(),
	   qy.getLocalArray()->ptrDev(),
	   qz.getLocalArray()->ptrDev());
      }
      else {
	computeQKernelNoExponential<<<blocksPerGrid,nThreads>>> 
	  (d, nbCells, m_weight[d],
	   cellFaces->getPtr()->ptrDev(),
	   m_faceCell.ptrDev(),
	   m_nbFacesInCell.ptrDev(),
	   isOutward.getLocalArray()->ptrDev(),
	   m_advanceOrder[d].ptrDev(),
	   volumes.getLocalArray()->ptrDev(),
	   m_fieldSource.ptrDev(),
	   m_fieldAbSrcV.ptrDev(),
	   m_fieldAbV.ptrDev(),
	   m_dotProdInFace.ptrDev(),
	   m_dirs.ptrDev(),
	   m_In.ptrDev(), m_II.ptrDev(), 
	   divQ.getLocalArray()->ptrDev(),
	   qx.getLocalArray()->ptrDev(),
	   qy.getLocalArray()->ptrDev(),
	   qz.getLocalArray()->ptrDev());
	   }*/
      CFLog(INFO, ")\n");
    }
  }
  
  socket_divq.getDataHandle().getLocalArray()->get();
  socket_qx.getDataHandle().getLocalArray()->get();
  socket_qy.getDataHandle().getLocalArray()->get();
  socket_qz.getDataHandle().getLocalArray()->get();
  
  CFLog(VERBOSE, "RadiativeTransferFVDOMCUDA::loopOverDirs() =>END\n");
}

//////////////////////////////////////////////////////////////////////////////

    } // namespace RadiativeTransfer

} // namespace COOLFluiD

//////////////////////////////////////////////////////////////////////////////

