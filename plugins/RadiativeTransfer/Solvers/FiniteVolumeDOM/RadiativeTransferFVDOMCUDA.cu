#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>

#include "Common/PE.hh"
#include "Common/BadValueException.hh"
#include "Common/CFPrintContainer.hh"
#include "Framework/CudaDeviceManager.hh"
#include "Common/CUDA/CFVec.hh"
#include "Framework/CudaTimer.hh"

#include "MathTools/MathConsts.hh"

#include "Environment/ObjectProvider.hh"
#include "Environment/CFEnv.hh"
#include "Environment/FileHandlerOutput.hh"
#include "Environment/FileHandlerInput.hh"
#include "Environment/DirPaths.hh"
#include "Environment/SingleBehaviorFactory.hh"

#include "Framework/PathAppender.hh"
#include "Framework/DataProcessing.hh"
#include "Framework/SubSystemStatus.hh"
#include "Framework/MethodCommandProvider.hh"
#include "Framework/MeshData.hh"
#include "Framework/PhysicalChemicalLibrary.hh"
#include "Framework/SocketBundleSetter.hh"

#include "FiniteVolume/CellCenterFVM.hh"

#include "RadiativeTransfer/RadiativeTransfer.hh"
#include "RadiativeTransfer/Solvers/FiniteVolumeDOM/RadiativeTransferFVDOMCUDA.hh"
#include "RadiativeTransfer/RadiationLibrary/Radiator.hh"
#include "RadiativeTransfer/RadiationLibrary/RadiationPhysicsHandler.hh"

// the following depends on the GPU model, should be user-defined
#define SHARED_ARRAY_SIZE 1536 

/////////////////////////////////////////////////////////////////////////////

using namespace std;
using namespace COOLFluiD::Framework;
using namespace COOLFluiD::Common;
using namespace COOLFluiD::MathTools;
using namespace COOLFluiD::Numerics::FiniteVolume;

//////////////////////////////////////////////////////////////////////////////

namespace COOLFluiD {

    namespace RadiativeTransfer {

//////////////////////////////////////////////////////////////////////////////

MethodCommandProvider<RadiativeTransferFVDOMCUDA, 
		      DataProcessingData, 
		      RadiativeTransferModule>
radiativeTransferFVDOMCUDAProvider("RadiativeTransferFVDOMCUDA");

//////////////////////////////////////////////////////////////////////////////

//this useful code sample is taken from 
 //http://stackoverflow.com/questions/12626096/why-has-atomicadd-not-been-implemented-for-doubles
inline __device__ double atomicAdd(double* address, double val)
{
  unsigned long long int* address_as_ull =
    (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
		    __double_as_longlong(val +
					 __longlong_as_double(assumed)));
  } while (assumed != old);
  return __longlong_as_double(old);
}

//////////////////////////////////////////////////////////////////////////////

__global__ void getFieldOpacitiesKernel(const bool useExponentialMethod,
					const CFuint TID, 
					const CFuint PID,
					const CFuint nbTemp,
					const CFuint nbPress,
					const CFuint nbBins,
					const CFuint ib,
					const CFuint nbEqs,
					const CFuint nbCells,
					const CFreal* Ttable,
					const CFreal* Ptable,
					const CFreal* states,
					const CFreal* volumes,
					const CFreal* opacities,
					const CFreal* radSource,
					CFreal* fieldSource,
					CFreal* fieldAbsor,
					CFreal* fieldAbSrcV,
					CFreal* fieldAbV)
{    
  // each thread takes care of computing the gradient for one single cell
  const int cellID = threadIdx.x + blockIdx.x*blockDim.x;
  
  if (cellID < nbCells) {
    fieldSource[cellID] = 0.;
    if(useExponentialMethod) {
      fieldAbsor[cellID]  = 0.;
    }
    else {
      fieldAbSrcV[cellID] = 0.;
      fieldAbV[cellID]    = 0.;
    }
    
    //Get the field pressure and T commented because now we impose a temperature profile
    const CFuint sIdx = cellID*nbEqs; 
    const CFreal p = states[sIdx + PID];
    const CFreal T = states[sIdx + TID];
    const CFreal patm = p/101325.; //converting from Pa to atm
    
    CFreal val1 = 0;
    CFreal val2 = 0;
    
    RadiativeTransferFVDOM::DeviceFunc interp;
    interp.tableInterpolate(nbBins, nbTemp, nbPress, Ttable, Ptable,
			    opacities, radSource, T, patm, ib, val1, val2); 
    
    if(useExponentialMethod){
      if (val1 <= 1e-30 || val2 <= 1e-30 ){
	fieldSource[cellID] = 1e-30;
	fieldAbsor[cellID]  = 1e-30;
      }
      else {
	fieldSource[cellID] = val2/val1;
	fieldAbsor[cellID]  = val1;
      }
    }
    else{
      if (val1 <= 1e-30 || val2 <= 1e-30 ){
	fieldSource[cellID] = 1e-30;
	fieldAbV[cellID]    = 1e-30*volumes[cellID]; // Volume converted from m^3 into cm^3
      }
      else {
	fieldSource[cellID] = val2/val1;
	fieldAbV[cellID]    = val1*volumes[cellID];
      }      
      fieldAbSrcV[cellID]   = fieldSource[cellID]*fieldAbV[cellID];
    }
  }
}

//////////////////////////////////////////////////////////////////////////////

__global__ void computeQKernelExponentialDirBigMem(const CFuint dStart,
						   const CFuint dEnd,
						   const CFuint nbCells,
						   const CFuint* cellFaces,
						   const CFint* faceCell,
						   const CFuint* nbFacesInCell,
						   const CFint* isOutward,
						   const CFint* advanceOrder,
						   const CFreal* weightIn,
						   const CFreal* volumes,
						   const CFreal* fieldSource,
						   const CFreal* fieldAbsor,
						   const CFreal* normals,
						   const CFreal* mdirs,
						   CFreal* In,
						   CFreal* divq,
						   CFreal* qx, CFreal* qy, CFreal* qz)
{ 
  // mdirs is used often, by all threads and should be shared
  //  __shared__ CFreal mdirs[256]; // overallocated memory
  const CFuint nbDirs = dEnd - dStart; 
  //CFint tID = threadIdx.x; 
  //while (tID < nbDirs) {
  //  mdirs[dStart+tID] = mdirsIn[dStart+tID];
  //  tID += blockDim.x;
  // }
  // __syncthreads();
  
  // each thread takes care of computing the gradient for one single cell
  const CFuint dirID = threadIdx.x + blockIdx.x*blockDim.x;
  if (dirID < nbDirs) {
    RadiativeTransferFVDOM::DeviceFunc fun;
    const CFuint d = dStart + dirID;
    const CFreal weight = weightIn[d];
    const CFuint d3 = d*3;
    const CFreal mdirs30 = mdirs[d3];
    const CFreal mdirs31 = mdirs[d3+1];
    const CFreal mdirs32 = mdirs[d3+2]; 
    const CFuint startCell = dirID*nbCells;
    for (CFuint m = 0; m < nbCells; ++m) {
      CFreal inDirDotnANeg = 0.;
      CFreal Ic            = 0.;
      CFreal dirDotnANeg   = 0.;
      CFreal Lc            = 0.;
      CFreal halfExp       = 0.;
      
      // allocate the cell entity
      const CFuint iCell   = abs(advanceOrder[startCell+m]);
      const CFuint nbFaces = nbFacesInCell[iCell];
      for (CFuint iFace = 0; iFace < nbFaces; ++iFace) { 
	const CFuint faceID = cellFaces[iFace*nbCells + iCell];
	const CFreal factor = ((CFuint)(isOutward[faceID]) != iCell) ? -1. : 1.;
	const CFuint startID = faceID*3;
	const CFreal dotProdInFace = fun.getDirDotNA(d,&mdirs[0],&normals[startID]);
	const CFreal dirDotNA = dotProdInFace*factor;
	
	if(dirDotNA < 0.) {
	  dirDotnANeg += dirDotNA;
	  const CFint fcellID = faceCell[faceID*2]; 
	  const CFint neighborCellID = (fcellID == iCell) ? faceCell[faceID*2+1] : fcellID;
	  const CFreal source = (neighborCellID >=0) ? In[neighborCellID*nbDirs+dirID] : fieldSource[iCell];
	  inDirDotnANeg += source*dirDotNA;
	   
	  /*if (iCell==100 && dirID == 0) {
	    printf ("source   : %6.6f \n", source);
	    printf ("dirDotNA : %6.6f  \n", dirDotNA);
	    printf ("inDirDotnANeg : %6.6f \n",inDirDotnANeg);
	    printf ("factor   : %6.6f  \n", factor);
	    }*/
	}
      } 
      
      Lc        = volumes[iCell]/(- dirDotnANeg); 
      halfExp   = exp(-0.5*Lc*fieldAbsor[iCell]);
      const CFreal InCell = (inDirDotnANeg/dirDotnANeg)*halfExp*halfExp + 
	(1. - halfExp*halfExp)*fieldSource[iCell];
      Ic = (inDirDotnANeg/dirDotnANeg)*halfExp + (1. - halfExp)*fieldSource[iCell];

      CFreal inDirDotnA = inDirDotnANeg;
      for (CFuint iFace = 0; iFace < nbFaces; ++iFace) {
	const CFuint faceID = cellFaces[iFace*nbCells + iCell];
	const CFreal factor = ((CFuint)(isOutward[faceID]) != iCell) ? -1. : 1.;
	const CFuint startID = faceID*3;
	const CFreal dotProdInFace = fun.getDirDotNA(d,&mdirs[0],&normals[startID]);
	const CFreal dirDotNA = dotProdInFace*factor;
	if (dirDotNA > 0.) {
	  inDirDotnA += InCell*dirDotNA;
	}
      }
      
      const CFuint cellIDin = iCell*nbDirs + dirID; // dirID*nbCells+iCell;
      In[cellIDin] = InCell; 
      const CFreal IcWeight = Ic*weight;
      
      // here no atomics are needed since qi/divq is a (logically) 2D array, with one qi/diq array for each threads
      qx[cellIDin]    += mdirs30*IcWeight;
      qy[cellIDin]    += mdirs31*IcWeight;
      qz[cellIDin]    += mdirs32*IcWeight;
      divq[cellIDin]  += inDirDotnA*weight;
     
      /*if (iCell==100 && dirID == 0) {
        printf ("IcWeight    : %6.6f \n", IcWeight);
        printf ("inDirDotnA  : %6.6f \n",inDirDotnA);
        printf ("InCell      : %6.6f \n", InCell);
        printf ("cellIDin    : %d  \n", cellIDin);
        const CFreal qxIcell = qx[cellIDin];
        printf ("qx[iCell]   : %6.6f  \n", qxIcell);
        const CFreal divqIcell = divq[cellIDin];
        printf ("divq[iCell] : %6.6f  \n", divqIcell);
        const CFreal In0 = In[cellIDin];
        printf ("In[iCell]   : %6.6f  \n", In0);
        printf ("d3          : %d  \n", d3);
        printf ("mdirs[d3]   : %6.6f  \n", mdirs30);
        printf ("mdirs[d3+1] : %6.6f  \n", mdirs31);
        printf ("mdirs[d3+2] : %6.6f  \n", mdirs32);
      }*/
    }
  }
}
      
//////////////////////////////////////////////////////////////////////////////

__global__ void computeQKernelExponentialDirAtomic(const CFuint dStart,
						   const CFuint dEnd,
						   const CFuint nbCells,
						   const CFuint* cellFaces,
						   const CFint* faceCell,
						   const CFuint* nbFacesInCell,
						   const CFint* isOutward,
						   const CFint* advanceOrder,
						   const CFreal* weightIn,
						   const CFreal* volumes,
						   const CFreal* fieldSource,
						   const CFreal* fieldAbsor,
						   const CFreal* normals,
						   const CFreal* mdirs,
						   CFreal* In,
						   CFreal* divq,
						   CFreal* qx, CFreal* qy, CFreal* qz)
{ 
  // mdirs is used often, by all threads and should be shared 
  //__shared__ CFreal mdirs[256]; // overallocated memory
  const CFuint nbDirs = dEnd - dStart; 
 // CFint tID = threadIdx.x; 
 // while (tID < nbDirs) {
 //   mdirs[dStart+tID] = mdirsIn[dStart+tID];
 //   tID += blockDim.x;
 // }
 // __syncthreads();
  
  // each thread takes care of computing the gradient for one single cell
  const CFuint dirID = threadIdx.x + blockIdx.x*blockDim.x;
  if (dirID < nbDirs) {
    RadiativeTransferFVDOM::DeviceFunc fun;
    const CFuint d = dStart + dirID;
    const CFuint d3 = d*3;
    const CFreal mdirs30 = mdirs[d3];
    const CFreal mdirs31 = mdirs[d3+1];
    const CFreal mdirs32 = mdirs[d3+2];
    const CFreal weight = weightIn[d];
    const CFuint startCell = dirID*nbCells;
     
    for (CFuint m = 0; m < nbCells; ++m) {
      CFreal inDirDotnANeg = 0.;
      CFreal Ic            = 0.;
      CFreal dirDotnANeg   = 0.;
      CFreal Lc            = 0.;
      CFreal halfExp       = 0.;
     
      // allocate the cell entity
      const CFuint iCell   = abs(advanceOrder[startCell+m]);
      const CFuint nbFaces = nbFacesInCell[iCell];
      for (CFuint iFace = 0; iFace < nbFaces; ++iFace) { 
	const CFuint faceID = cellFaces[iFace*nbCells + iCell];
	const CFreal factor = ((CFuint)(isOutward[faceID]) != iCell) ? -1. : 1.;
	const CFuint startID = faceID*3;
	const CFreal dotProdInFace = fun.getDirDotNA(d,&mdirs[0],&normals[startID]);
	const CFreal dirDotNA = dotProdInFace*factor;
	
	if(dirDotNA < 0.) {
	  dirDotnANeg += dirDotNA;
	  const CFint fcellID = faceCell[faceID*2]; 
	  const CFint neighborCellID = (fcellID == iCell) ? faceCell[faceID*2+1] : fcellID;
	  const CFreal source = (neighborCellID >=0) ? In[neighborCellID*nbDirs+dirID] : fieldSource[iCell];
	  inDirDotnANeg += source*dirDotNA;
	  
	  /*if (iCell==100 && dirID == 0) {
	    printf ("source   : %6.6f \n", source);
	    printf ("dirDotNA : %6.6f  \n", dirDotNA);
	    printf ("inDirDotnANeg : %6.6f \n",inDirDotnANeg);
	    printf ("factor   : %6.6f  \n", factor);
	    }*/
	  
	}
      } 
      
      Lc        = volumes[iCell]/(- dirDotnANeg); 
      halfExp   = exp(-0.5*Lc*fieldAbsor[iCell]);
      const CFreal InCell = (inDirDotnANeg/dirDotnANeg)*halfExp*halfExp + 
	(1. - halfExp*halfExp)*fieldSource[iCell];
      Ic = (inDirDotnANeg/dirDotnANeg)*halfExp + (1. - halfExp)*fieldSource[iCell];
      
      CFreal inDirDotnA = inDirDotnANeg;
      for (CFuint iFace = 0; iFace < nbFaces; ++iFace) {
	const CFuint faceID = cellFaces[iFace*nbCells + iCell];
	const CFreal factor = ((CFuint)(isOutward[faceID]) != iCell) ? -1. : 1.;
	const CFuint startID = faceID*3;
	const CFreal dotProdInFace = fun.getDirDotNA(d,&mdirs[0],&normals[startID]);
	const CFreal dirDotNA = dotProdInFace*factor;
	if (dirDotNA > 0.) {
	  inDirDotnA += InCell*dirDotNA;
	}
      }
      
      const CFuint cellIDin = iCell*nbDirs + dirID; // dirID*nbCells+iCell;
      In[cellIDin] = InCell;
      const CFreal IcWeight = Ic*weight;
      const CFreal m0w = mdirs30*IcWeight;
      const CFreal m1w = mdirs31*IcWeight;
      const CFreal m2w = mdirs32*IcWeight;
      const CFreal inw = inDirDotnA*weight;    
  
      // here atomics are needed since qi/divq are single arrays shared by all threads
      atomicAdd(&qx[iCell], m0w);
      atomicAdd(&qy[iCell], m1w);          
      atomicAdd(&qz[iCell], m2w);
      atomicAdd(&divq[iCell], inw);
      
      /*if (iCell==100 && dirID == 0) {
	printf ("IcWeight    : %6.6f \n", IcWeight);
	printf ("inDirDotnA  : %6.6f \n",inDirDotnA);
	printf ("InCell      : %6.6f \n", InCell);
      	printf ("cellIDin    : %d  \n", cellIDin);
	const CFreal qxIcell = qx[iCell];
	printf ("qx[iCell]   : %6.6f  \n", qxIcell);
	const CFreal divqIcell = divq[iCell];
	printf ("divq[iCell] : %6.6f  \n", divqIcell);
	const CFreal In0 = In[cellIDin];
	printf ("In[iCell]   : %6.6f  \n", In0);
	printf ("d3          : %d  \n", d3);
	printf ("mdirs[d3]   : %6.6f  \n", mdirs30);
	printf ("mdirs[d3+1] : %6.6f  \n", mdirs31);
	printf ("mdirs[d3+2] : %6.6f  \n", mdirs32);
      }*/
    }
  }
}
      
//////////////////////////////////////////////////////////////////////////////

__global__ void computeQKernelExponentialDir(const CFuint dStart,
					     const CFuint dEnd,
					     const CFuint startCellID,
					     const CFuint endCellID,
					     const CFuint nbCells,
					     const CFuint* cellFaces,
					     const CFint* faceCell,
					     const CFuint* nbFacesInCell,
					     const CFint* isOutward,
					     const CFint* advanceOrder,
					     const CFreal* weightIn,
					     const CFreal* volumes,
					     const CFreal* fieldSource,
					     const CFreal* fieldAbsor,
					     const CFreal* normals,
					     const CFreal* mdirs,
					     CFreal* In,
					     CFreal* II,
					     CFreal* divq,
					     CFreal* qx, CFreal* qy, CFreal* qz)
{ 
  // mdirs is used often and should be shared 
  // __shared__ CFreal mdirs[2048]; // overallocated memory
  
  const CFuint nbDirs = dEnd - dStart; 
  // while (tID < nbDirs) {
  //   mdirs[dStart+tID] = mdirsIn[dStart+tID];
  //   tID += blockIdx.x*blockDim.x; // this could be buggy
  // }
  // __syncthreads();
  
  // this assumes 49,152 bytes of shared memory
  __shared__ CFreal qxSh[SHARED_ARRAY_SIZE];
  __shared__ CFreal qySh[SHARED_ARRAY_SIZE]; 
  __shared__ CFreal qzSh[SHARED_ARRAY_SIZE];
  __shared__ CFreal divqSh[SHARED_ARRAY_SIZE];
  
  CFint tID = threadIdx.x; 
  while (tID < SHARED_ARRAY_SIZE) {
    qxSh[tID] = qySh[tID] = qzSh[tID] = divqSh[tID] = 0.;
    tID += blockDim.x;
  }
  __syncthreads();
  
 // printf ("BBBBBBBB   : %d %d \n", blockDim.x, blockIdx.x);
  
  CFuint iCellList[SHARED_ARRAY_SIZE];
  
  // each thread takes care of computing the gradient for one single cell
  const CFuint dirID = threadIdx.x + blockIdx.x*blockDim.x;
  if (dirID < nbDirs) {
    RadiativeTransferFVDOM::DeviceFunc fun;
    const CFuint d = dStart + dirID;
    const CFreal weight = weightIn[d];
    const CFuint startCell = dirID*nbCells;
    CFuint iCellSh = 0;
    for (CFuint m = startCellID; m < endCellID; ++m, ++iCellSh) {
      CFreal inDirDotnANeg = 0.;
      CFreal Ic            = 0.;
      CFreal dirDotnANeg   = 0.;
      CFreal Lc            = 0.;
      CFreal halfExp       = 0.;
      
      // allocate the cell entity
      const CFuint iCell   = abs(advanceOrder[startCell+m]);
      iCellList[iCellSh] = iCell;
      const CFuint nbFaces = nbFacesInCell[iCell];
      for (CFuint iFace = 0; iFace < nbFaces; ++iFace) { 
	const CFuint faceID = cellFaces[iFace*nbCells + iCell];
	const CFreal factor = ((CFuint)(isOutward[faceID]) != iCell) ? -1. : 1.;
	const CFuint startID = faceID*3;
	const CFreal dotProdInFace = fun.getDirDotNA(d,&mdirs[0],&normals[startID]);
	const CFreal dirDotNA = dotProdInFace*factor;
	
	if(dirDotNA < 0.) {
	  dirDotnANeg += dirDotNA;
	  const CFint fcellID = faceCell[faceID*2]; 
	  const CFint neighborCellID = (fcellID == iCell) ? faceCell[faceID*2+1] : fcellID;
	  const CFreal source = (neighborCellID >=0) ? In[nbCells*dirID+neighborCellID] : fieldSource[iCell];
	  inDirDotnANeg += source*dirDotNA;
	  
	  /*if (iCell==100) {
	    printf ("source   : %6.6f \n", source);
	    printf ("dirDotNA : %6.6f  \n", dirDotNA);
	    printf ("inDirDotnANeg : %6.6f \n",inDirDotnANeg);
	    printf ("factor   : %6.6f  \n", factor);
	    }*/
	}
      } 
      const CFuint cellIDin = nbCells*dirID+iCell;
      Lc        = volumes[iCell]/(- dirDotnANeg); 
      halfExp   = exp(-0.5*Lc*fieldAbsor[iCell]);
      In[cellIDin] = (inDirDotnANeg/dirDotnANeg)*halfExp*halfExp + (1. - halfExp*halfExp)*fieldSource[iCell];
      Ic        = (inDirDotnANeg/dirDotnANeg)*halfExp + (1. - halfExp)*fieldSource[iCell];
      
      /*if (iCell==100) {
	printf ("Lc   : %6.6f  \n", Lc);
	printf ("halfExp : %6.6f  \n", halfExp);
	printf ("In : %6.6f \n",In[cellIDin]);
	printf ("Ic   : %6.6f  \n", Ic);
	printf ("weight   : %6.6f \n", weight);
	}*/

      CFreal inDirDotnA = inDirDotnANeg;
      for (CFuint iFace = 0; iFace < nbFaces; ++iFace) {
	const CFuint faceID = cellFaces[iFace*nbCells + iCell];
	const CFreal factor = ((CFuint)(isOutward[faceID]) != iCell) ? -1. : 1.;
	const CFuint startID = faceID*3;
	const CFreal dotProdInFace = fun.getDirDotNA(d,&mdirs[0],&normals[startID]);
	const CFreal dirDotNA = dotProdInFace*factor;
	if (dirDotNA > 0.) {
	  inDirDotnA += In[cellIDin]*dirDotNA;
	}
      }
      
      const CFreal IcWeight = Ic*weight;
      II[cellIDin] += IcWeight;
      const CFuint d3 = d*3;
      atomicAdd(&qxSh[iCellSh], mdirs[d3]*IcWeight);
      atomicAdd(&qySh[iCellSh], mdirs[d3+1]*IcWeight);          
      atomicAdd(&qzSh[iCellSh], mdirs[d3+2]*IcWeight);
      atomicAdd(&divqSh[iCellSh], inDirDotnA*weight);
    }
  }
  
  __syncthreads();
  
  tID = threadIdx.x; 
  const CFuint nbCellsSh = endCellID - startCellID;
  // printf ("start, end, nbCellsSh   : %d %d %d\n", startCellID, endCellID, nbCellsSh);
 
  // at this point should be iCellSh == nbCellsSh
  while (tID < nbCellsSh) {
    // the following could be done outside the loop
    const CFuint iCell = iCellList[tID];
    atomicAdd(&qx[iCell], qxSh[tID]);
    atomicAdd(&qy[iCell], qySh[tID]);
    atomicAdd(&qz[iCell], qzSh[tID]);
    atomicAdd(&divq[iCell], divqSh[tID]);
    tID += blockDim.x; 
  }
}
  
//////////////////////////////////////////////////////////////////////////////
 
__global__ void computeQKernelNoExponentialDir(const CFuint dStart,
					       const CFuint dEnd,
					       const CFuint nbCells,
					       const CFuint* cellFaces,
					       const CFint* faceCell,
					       const CFuint* nbFacesInCell,
					       const CFint* isOutward,
					       const CFint* advanceOrder,
					       const CFreal* weightIn,
					       const CFreal* volumes,
                                               const CFreal* fieldSource,
                                               const CFreal* fieldAbSrcV,
                                               const CFreal* fieldAbV,
					       const CFreal* normals,
					       const CFreal* mdirsIn,
					       CFreal* In,
					       //					       CFreal* II,
					       CFreal* divQ,
					       CFreal* qx, CFreal* qy, CFreal* qz)
{ 
  // mdirs is used often and should be shared 
  __shared__ CFreal mdirs[2048]; // overallocated memory
  CFint tID = threadIdx.x; 
  const CFuint nbDirs = dEnd - dStart; 
  while (tID < nbDirs) {
    mdirs[dStart+tID] = mdirsIn[dStart+tID];
    tID += blockDim.x; // this could be buggy
  }
  __syncthreads();
  
  // each thread takes care of computing the gradient for one single cell
  const CFuint dirID = threadIdx.x + blockIdx.x*blockDim.x;
  if (dirID < nbDirs) {
    RadiativeTransferFVDOM::DeviceFunc fun;
    const CFuint d = dStart + dirID;
    const CFreal weight = weightIn[d];
    const CFuint startCell = dirID*nbCells;
    for (CFuint m = 0; m < nbCells; ++m) {    
      // allocate the cell entity
      const CFuint iCell   = abs(advanceOrder[startCell+m]);
      CFreal inDirDotnANeg = 0.;
      CFreal Ic            = 0.;
      CFreal dirDotnAPos   = 0.;
      
      const CFuint nbFaces = nbFacesInCell[iCell];
      for (CFuint iFace = 0; iFace < nbFaces; ++iFace) {
	const CFuint faceID = cellFaces[iFace*nbCells + iCell];
	const CFreal factor = ((CFuint)(isOutward[faceID]) != iCell) ? -1. : 1.;
	const CFuint startID = faceID*3;
	const CFreal dotProdInFace = fun.getDirDotNA(d,&mdirs[0],&normals[startID]);
	const CFreal dirDotNA = dotProdInFace*factor;
	
	if (dirDotNA >= 0.){
	  dirDotnAPos += dirDotNA;
	}
	else {
	  const CFint fcellID = faceCell[faceID*2]; 
	  const CFint neighborCellID = (fcellID == iCell) ? faceCell[faceID*2+1] : fcellID;
	  const CFreal source = (neighborCellID >=0) ? In[nbCells*dirID+neighborCellID] : fieldSource[iCell];
	  inDirDotnANeg += source*dirDotNA;
	}
      } 
      const CFuint cellIDin = nbCells*dirID+iCell;
      In[cellIDin] = (fieldAbSrcV[iCell] - inDirDotnANeg)/(fieldAbV[iCell] + dirDotnAPos);
      Ic = In[cellIDin];
      
      qx[iCell] += Ic*mdirs[d*3]*weight;
      qy[iCell] += Ic*mdirs[d*3+1]*weight;
      qz[iCell] += Ic*mdirs[d*3+2]*weight;
      
      CFreal inDirDotnA = inDirDotnANeg;
      for (CFuint iFace = 0; iFace < nbFaces; ++iFace) {
	const CFuint faceID = cellFaces[iFace*nbCells + iCell];
	const CFreal factor = ((CFuint)(isOutward[faceID]) != iCell) ? -1. : 1.;
	const CFuint startID = faceID*3;
	const CFreal dotProdInFace = fun.getDirDotNA(d,&mdirs[0],&normals[startID]);
	const CFreal dirDotNA = dotProdInFace*factor;
	if (dirDotNA > 0.) {
	  inDirDotnA += In[cellIDin]*dirDotNA;
	}
      }
      
      divQ[iCell] += inDirDotnA*weight;
      //  II[nbCells*dirID+iCell] += Ic*weight;
    }  
  }
}
      
//////////////////////////////////////////////////////////////////////////////
 
__global__ void computeQKernelExponential(const CFuint d,
					  const CFuint nbCells,
					  const CFreal weightIn,
					  const CFuint* cellFaces,
					  const CFint* faceCell,
					  const CFuint* nbFacesInCell,
					  const CFint* isOutward,
					  const CFint* advanceOrder,
					  const CFreal* volumes,
					  const CFreal* fieldSource,
					  const CFreal* fieldAbsor,
					  const CFreal* dotProdInFace,
					  const CFreal* mdirs,
					  CFreal* In,
					  CFreal* II,
					  CFreal* divQ,
					  CFreal* qx, CFreal* qy, CFreal* qz)
{    
  // each thread takes care of computing the gradient for one single cell
  const int cellID = threadIdx.x + blockIdx.x*blockDim.x;
  
  __shared__ CFreal weight;
  weight = weightIn;
  __syncthreads();
  
  if (cellID < nbCells) {
    CFreal inDirDotnANeg = 0.;
    CFreal Ic            = 0.;
    CFreal dirDotnANeg   = 0.;
    CFreal Lc            = 0.;
    CFreal halfExp       = 0.;
        
    // allocate the cell entity
    const CFuint iCell   = abs(advanceOrder[d*nbCells+cellID]);
    
    const CFuint nbFaces = nbFacesInCell[iCell];
    for (CFuint iFace = 0; iFace < nbFaces; ++iFace) { 
      const CFuint faceID = cellFaces[iFace*nbCells + iCell];
      const CFreal factor = ((CFuint)(isOutward[faceID]) != iCell) ? -1. : 1.;
      const CFreal dirDotNA = dotProdInFace[faceID]*factor;
      
      if(dirDotNA < 0.) {
	dirDotnANeg += dirDotNA;
	
        const CFint fcellID = faceCell[faceID*2]; 
        const CFint neighborCellID = (fcellID == iCell) ? faceCell[faceID*2+1] : fcellID;
	const CFreal source = (neighborCellID >=0) ? In[neighborCellID] : fieldSource[iCell];
        inDirDotnANeg += source*dirDotNA;
      

	if (iCell==100) {
	  printf ("source   : %6.6f \n", source);
	  printf ("dirDotNA : %6.6f  \n", dirDotNA);
	  printf ("inDirDotnANeg : %6.6f \n",inDirDotnANeg);
	  printf ("factor   : %6.6f  \n", factor);
	}
	
	
      }
    } 
    Lc        = volumes[iCell]/(- dirDotnANeg); 
    halfExp   = exp(-0.5*Lc*fieldAbsor[iCell]);
    In[iCell] = (inDirDotnANeg/dirDotnANeg)*halfExp*halfExp + (1. - halfExp*halfExp)*fieldSource[iCell];
    Ic        = (inDirDotnANeg/dirDotnANeg)*halfExp + (1. - halfExp)*fieldSource[iCell];
    
    if (iCell==100) {
      printf ("Lc   : %6.6f  \n", Lc);
      printf ("halfExp : %6.6f  \n", halfExp);
      printf ("In : %6.6f \n",In[iCell]);
      printf ("Ic   : %6.6f  \n", Ic);
      printf ("weight   : %6.6f \n", weight);
    }
    
    qx[iCell] += Ic*mdirs[d*3]*weight;
    qy[iCell] += Ic*mdirs[d*3+1]*weight;
    qz[iCell] += Ic*mdirs[d*3+2]*weight;
    
    CFreal inDirDotnA = inDirDotnANeg;
    for (CFuint iFace = 0; iFace < nbFaces; ++iFace) {
      const CFuint faceID = cellFaces[iFace*nbCells + iCell];
      const CFreal factor = ((CFuint)(isOutward[faceID]) != iCell) ? -1. : 1.;
      const CFreal dirDotNA = dotProdInFace[faceID]*factor;
      if (dirDotNA > 0.) {
	inDirDotnA += In[iCell]*dirDotNA;
      }
    }
    
    divQ[iCell] += inDirDotnA*weight;
    II[iCell]   += Ic*weight;
  }  
}
      
//////////////////////////////////////////////////////////////////////////////

__global__ void computeQKernelNoExponential(const CFuint d, 
					    const CFuint nbCells,
					    const CFreal weightIn,
					    const CFuint* cellFaces,
					    const CFint* faceCell,
					    const CFuint* nbFacesInCell,
					    const CFint* isOutward,
					    const CFint* advanceOrder,
					    const CFreal* volumes,
					    const CFreal* fieldSource,
					    const CFreal* fieldAbSrcV,
					    const CFreal* fieldAbV,
					    const CFreal* dotProdInFace,
					    const CFreal* mdirs,
					    CFreal* In,
					    CFreal* II,
					    CFreal* divQ,
					    CFreal* qx, CFreal* qy, CFreal* qz)
{    
  // each thread takes care of computing the gradient for one single cell
  const int cellID = threadIdx.x + blockIdx.x*blockDim.x;
  
  __shared__ CFreal weight;
  weight = weightIn;
  __syncthreads();

  if (cellID < nbCells) {
    
    // allocate the cell entity
    const CFuint iCell = abs(advanceOrder[d*nbCells+cellID]);
    CFreal inDirDotnANeg = 0.;
    CFreal Ic            = 0.;
    CFreal dirDotnAPos   = 0.;

    const CFuint nbFaces = nbFacesInCell[iCell];
    for (CFuint iFace = 0; iFace < nbFaces; ++iFace) {
      const CFuint faceID = cellFaces[iFace*nbCells + iCell];
      const CFreal factor = ((CFuint)(isOutward[faceID]) != iCell) ? -1. : 1.;
      const CFreal dirDotNA = dotProdInFace[faceID]*factor;
      
      if (dirDotNA >= 0.){
	dirDotnAPos += dirDotNA;
      }
      else {
	const CFint fcellID = faceCell[faceID*2]; 
        const CFint neighborCellID = (fcellID == iCell) ? faceCell[faceID*2+1] : fcellID;
	const CFreal source = (neighborCellID >=0) ? In[neighborCellID] : fieldSource[iCell];
        inDirDotnANeg += source*dirDotNA;
      }
    } 
    In[iCell] = (fieldAbSrcV[iCell] - inDirDotnANeg)/(fieldAbV[iCell] + dirDotnAPos);
    Ic = In[iCell];
    
    qx[iCell] += Ic*mdirs[d*3]*weight;
    qy[iCell] += Ic*mdirs[d*3+1]*weight;
    qz[iCell] += Ic*mdirs[d*3+2]*weight;
    
    CFreal inDirDotnA = inDirDotnANeg;
    for (CFuint iFace = 0; iFace < nbFaces; ++iFace) {
      const CFuint faceID = cellFaces[iFace*nbCells + iCell];
      const CFreal factor = ((CFuint)(isOutward[faceID]) != iCell) ? -1. : 1.;
      const CFreal dirDotNA = dotProdInFace[faceID]*factor;
      if (dirDotNA > 0.) {
	inDirDotnA += In[iCell]*dirDotNA;
      }
    }
    
    divQ[iCell] += inDirDotnA*weight;
    II[iCell] += Ic*weight;
  }  
}
      
//////////////////////////////////////////////////////////////////////////////

RadiativeTransferFVDOMCUDA::RadiativeTransferFVDOMCUDA(const std::string& name) :
  RadiativeTransferFVDOM(name),
  m_faceCell(),
  m_nbFacesInCell(),
  m_InDir(),
  m_qxDir(),
  m_qyDir(),
  m_qzDir(),
  m_divqDir()  
{
  addConfigOptionsTo(this);
  
  m_qAlgoName = "Atomic";
  setParameter("QAlgoName", &m_qAlgoName);
}
      
//////////////////////////////////////////////////////////////////////////////

RadiativeTransferFVDOMCUDA::~RadiativeTransferFVDOMCUDA()
{
}
      
//////////////////////////////////////////////////////////////////////////////

void RadiativeTransferFVDOMCUDA::defineConfigOptions(Config::OptionList& options)
{  
  options.addConfigOption< string >
    ("QAlgoName",
     "Name of the algorithm (Atomic, BigMem) to use for computing Q on the GPU.");
}
      
//////////////////////////////////////////////////////////////////////////////

void RadiativeTransferFVDOMCUDA::setup()
{
  CFAUTOTRACE;
    
  RadiativeTransferFVDOM::setup();
  
  CFLog(VERBOSE, "RadiativeTransferFVDOMCUDA::setup() => START\n");
  
  // store invariant data on GPU
  SafePtr<ConnectivityTable<CFuint> > cellFaces = 
    MeshDataStack::getActive()->getConnectivity("cellFaces");
  cellFaces->getPtr()->put();
  m_dirs.put();
  m_weight.put();
  
  /*CFLog(INFO, "dirs: ");
  for (CFuint i = 0; i < m_dirs.size(); ++i) {
   CFLog(INFO, m_dirs[i] << " "); 
  }

  CFLog(INFO, "weights: ");
  for (CFuint i = 0; i < m_weight.size(); ++i) {
   CFLog(INFO, m_weight[i] << " ");  
  }*/

  m_fieldSource.put();  // to be removed 
  m_fieldAbsor.put();  // to be removed
  m_fieldAbSrcV.put(); // to be removed
  m_fieldAbV.put();  // to be removed
  if (!m_loopOverBins) {
    m_In.put(); // to be removed
    m_II.put(); // to be removed
  }
  m_opacities.put();
  m_radSource.put();
  m_Ttable.put();
  m_Ptable.put();
  m_advanceOrder.put(); // this can be a very big storage
  
  // AL: redundant
  const CFuint nbCells = m_fieldSource.size();
  if(m_useExponentialMethod){
    for (CFuint i=0;i<nbCells;++i) {
      m_fieldSource[i] = m_fieldAbsor[i] = m_In[i] = m_II[i] = 0;
    }
  }
  else {
    for (CFuint i=0;i<nbCells;++i) {
      m_fieldSource[i] = m_fieldAbSrcV[i] = m_fieldAbV[i] = m_In[i] = m_II[i] = 0;
    }
  }
  
  const CFuint totalNbFaces = MeshDataStack::getActive()->Statistics().getNbFaces();
  m_faceCell.resize(totalNbFaces*2);
  m_faceCell = -1;
    
  m_nbFacesInCell.resize(nbCells);
  for (CFuint iCell = 0; iCell < nbCells; ++iCell) {
    const CFuint nbFaces = cellFaces->nbCols(iCell);
    m_nbFacesInCell[iCell] = nbFaces;
    for (CFuint iFace = 0; iFace < nbFaces; ++iFace) {
      const CFuint faceID2 = (*cellFaces)(iCell, iFace)*2;
      if (m_faceCell[faceID2] == -1) {
	m_faceCell[faceID2] = iCell;
      }
      else {
	m_faceCell[faceID2+1] = iCell;
      }
    }
  }
  
  m_faceCell.put();
  m_nbFacesInCell.put();
  
  if (m_loopOverBins) {  
    // AL: check if this creates a memory leak at the exit when !m_loopOverBins
    const CFuint nbDirs = m_advanceOrder.size()/nbCells;
    m_InDir.resize(nbCells*nbDirs);
    
    if (m_qAlgoName == "BigMem") {
      m_qxDir.resize(nbCells*nbDirs);
      m_qyDir.resize(nbCells*nbDirs);
      m_qzDir.resize(nbCells*nbDirs);
      m_divqDir.resize(nbCells*nbDirs);
      
      for (CFuint i = 0; i < m_InDir.size(); ++i) {
	m_qxDir[i] = m_qyDir[i] = m_qzDir[i] = m_divqDir[i] = 0.;
      }
      
      m_qxDir.put();
      m_qyDir.put();
      m_qzDir.put();
      m_divqDir.put();
    }
    
    for (CFuint i = 0; i < m_InDir.size(); ++i) {
      m_InDir[i] = 0.;
    }
    m_InDir.put();
  }
  
  CFLog(VERBOSE, "RadiativeTransferFVDOMCUDA::setup() => END\n");
}
      
//////////////////////////////////////////////////////////////////////////////

void RadiativeTransferFVDOMCUDA::unsetup()
{
  CFAUTOTRACE;
  
  RadiativeTransferFVDOM::unsetup();
}
      
//////////////////////////////////////////////////////////////////////////////
 
void RadiativeTransferFVDOMCUDA::loopOverDirs(const CFuint startBin, 
					      const CFuint endBin, 
					      const CFuint startDir,
					      const CFuint endDir)
{
  CFLog(VERBOSE, "RadiativeTransferFVDOMCUDA::loopOverDirs() => START\n");
  
  SafePtr<ConnectivityTable<CFuint> > cellFaces = 
    MeshDataStack::getActive()->getConnectivity("cellFaces");
  DataHandle<CFint> isOutward = socket_isOutward.getDataHandle();
  DataHandle<CFreal> volumes = socket_volumes.getDataHandle();
  DataHandle<CFreal> divQ = socket_divq.getDataHandle();
  DataHandle<State*, GLOBAL> states = socket_states.getDataHandle();
  DataHandle<CFreal> qx = socket_qx.getDataHandle();
  DataHandle<CFreal> qy = socket_qy.getDataHandle();
  DataHandle<CFreal> qz = socket_qz.getDataHandle();
  
  const CFuint nbCells = states.size();
  const CFuint nbEqs = PhysicalModelStack::getActive()->getNbEq();
  
  states.getGlobalArray()->put();
  isOutward.getLocalArray()->put(); 
  volumes.getLocalArray()->put(); 
  
  const CFuint blocksPerGrid = 
    CudaEnv::CudaDeviceManager::getInstance().getBlocksPerGrid(nbCells);
  const CFuint nThreads = CudaEnv::CudaDeviceManager::getInstance().getNThreads();
    
  for (CFuint d = startDir; d < endDir; ++d) {
    CFLog(INFO, "( dir: " << d << " ), ( bin: ");
    const CFuint bStart = (d != startDir) ? 0 : startBin;
    const CFuint bEnd   = (d != m_startEndDir.second) ? m_nbBins : endBin;
    
    // precompute dot products for all faces and directions (a part from the sign)
    computeDotProdInFace(d, m_dotProdInFace);
    m_dotProdInFace.put();
    
    for (CFuint ib = startBin; ib < endBin; ++ib) {
      CFLog(INFO, ib << " ");
      
      // precompute the radiation properties for all cells
      getFieldOpacitiesKernel<<<blocksPerGrid,nThreads>>>
	(m_useExponentialMethod, 
	 m_TID, m_PID, m_nbTemp, m_nbPress, m_nbBins,
	 ib, nbEqs, nbCells, m_Ttable.ptrDev(), m_Ptable.ptrDev(), 
	 states.getGlobalArray()->ptrDev(),
	 volumes.getLocalArray()->ptrDev(),
	 m_opacities.ptrDev(),
	 m_radSource.ptrDev(),
	 m_fieldSource.ptrDev(),
	 m_fieldAbsor.ptrDev(),
	 m_fieldAbSrcV.ptrDev(),
	 m_fieldAbV.ptrDev());  
      
      m_fieldSource.get();
      m_fieldAbsor.get();
      m_fieldAbSrcV.get();
      m_fieldAbV.get();
      
      // compute the radiative heat flux: unfortunately, this cannot be done
      // on the GPU since the computeQ algorithm is serial (the result on one 
      // cell depends on previously computed cells) and must follow the predefined 
      // advance ordering
      (m_useExponentialMethod) ? 
	computeQExponential(ib,startDir,d) : computeQNoExponential(ib,startDir,d);
    }
    CFLog(INFO, ")\n");
  }
  
  CFLog(VERBOSE, "RadiativeTransferFVDOMCUDA::loopOverDirs() =>END\n");
}

//////////////////////////////////////////////////////////////////////////////
      
void RadiativeTransferFVDOMCUDA::loopOverBins(const CFuint startBin, 
					      const CFuint endBin, 
					      const CFuint startDir,
					      const CFuint endDir)
{
  CFLog(VERBOSE, "RadiativeTransferFVDOMCUDA::loopOverBins() => START\n");
  
  SafePtr<ConnectivityTable<CFuint> > cellFaces = 
    MeshDataStack::getActive()->getConnectivity("cellFaces");
  DataHandle<CFint> isOutward = socket_isOutward.getDataHandle();
  DataHandle<CFreal> volumes = socket_volumes.getDataHandle();
  DataHandle<CFreal> divQ = socket_divq.getDataHandle();
  DataHandle<State*, GLOBAL> states = socket_states.getDataHandle();
  DataHandle<CFreal> qx = socket_qx.getDataHandle();
  DataHandle<CFreal> qy = socket_qy.getDataHandle();
  DataHandle<CFreal> qz = socket_qz.getDataHandle();
  DataHandle<CFreal> normals = socket_normals.getDataHandle();
  
  const CFuint nbCells = states.size();
  const CFuint nbEqs = PhysicalModelStack::getActive()->getNbEq();
  
  // store useful data on the GPU
  states.getGlobalArray()->put();
  isOutward.getLocalArray()->put(); 
  volumes.getLocalArray()->put(); 
  socket_normals.getDataHandle().getLocalArray()->put();
  
  const CFuint blocksPerGrid = 
    CudaEnv::CudaDeviceManager::getInstance().getBlocksPerGrid(nbCells);
  const CFuint nThreads = CudaEnv::CudaDeviceManager::getInstance().getNThreads();
  
  // if more than one iteration is needed, the initialization has to be done here
  // for the moment it is done the RadiativeTransferFVDOM::setup();
  // divq = 0.; qx = 0.; qy = 0.; qz = 0.; 
  socket_divq.getDataHandle().getLocalArray()->put();
  socket_qx.getDataHandle().getLocalArray()->put();
  socket_qy.getDataHandle().getLocalArray()->put();
  socket_qz.getDataHandle().getLocalArray()->put();

  Stopwatch<WallTime> stp;
  stp.start();
  
  for(CFuint ib = startBin; ib < endBin; ++ib) {
    CFLog(INFO, "( bin: " << ib << " ), ( dir: ");
    
    if (m_oldAlgo) { 
      // precompute the radiation properties for all cells
      getFieldOpacitiesKernel<<<blocksPerGrid,nThreads>>>
	(m_useExponentialMethod, 
	 m_TID, m_PID, m_nbTemp, m_nbPress, m_nbBins,
	 ib, nbEqs, nbCells, m_Ttable.ptrDev(), m_Ptable.ptrDev(), 
	 states.getGlobalArray()->ptrDev(),
	 volumes.getLocalArray()->ptrDev(),
	 m_opacities.ptrDev(),
	 m_radSource.ptrDev(),
	 m_fieldSource.ptrDev(),
	 m_fieldAbsor.ptrDev(),
	 m_fieldAbSrcV.ptrDev(),
	 m_fieldAbV.ptrDev());  
    }
    
    /* m_fieldSource.get();
       m_fieldAbsor.get();
       m_fieldAbSrcV.get();
       m_fieldAbV.get();
       
       for (CFuint k = 0; k < m_fieldSource.size(); ++k)
       CFLog(INFO, "m_fieldSource[" <<k << "] => (" << m_fieldSource[k] << "\n");
       for (CFuint k = 0; k < m_fieldAbsor.size(); ++k)
       CFLog(INFO, "m_fieldAbsor[" <<k << "] => (" << m_fieldAbsor[k] << "\n");
       for (CFuint k = 0; k < m_fieldAbSrcV.size(); ++k)
       CFLog(INFO, "m_fieldAbSrcV[" <<k << "] => (" << m_fieldAbSrcV[k] << "\n");
       for (CFuint k = 0; k < m_fieldAbV.size(); ++k)
       CFLog(INFO, "m_fieldAbV[" <<k << "] => (" << m_fieldAbV[k] << "\n");
       
       AL: this works perfectly
       exit(1);*/
    
    const CFuint dStart = (ib != startBin) ? 0 : startDir;
    const CFuint dEnd = (ib != m_startEndBin.second)? m_nbDirs : endDir;
    const CFuint nbDirs = dEnd - dStart;
    const CFuint blocksPerDir = 
      CudaEnv::CudaDeviceManager::getInstance().getBlocksPerGrid(nbDirs);
    const CFuint nThreadsDir = 
      std::min((CFuint)CudaEnv::CudaDeviceManager::getInstance().getNThreads(), nbDirs);
    
    if (m_useExponentialMethod) {
      if (m_qAlgoName == "Atomic") {
	// AL: this algorithm is my preferred choice, since needs much less memory
	//     but uses atomics
	computeQKernelExponentialDirAtomic<<<blocksPerDir,nThreadsDir>>> 
	  (dStart, dEnd, nbCells,
	   cellFaces->getPtr()->ptrDev(),
	   m_faceCell.ptrDev(),
	   m_nbFacesInCell.ptrDev(),
	   isOutward.getLocalArray()->ptrDev(),
	   m_advanceOrder.ptrDev(),
	   m_weight.ptrDev(),
	   volumes.getLocalArray()->ptrDev(),
	   m_fieldSource.ptrDev(),
	   m_fieldAbsor.ptrDev(),
	   normals.getLocalArray()->ptrDev(),
	   m_dirs.ptrDev(),
	   m_InDir.ptrDev(),
	   divQ.getLocalArray()->ptrDev(),
	   qx.getLocalArray()->ptrDev(),
	   qy.getLocalArray()->ptrDev(),
	   qz.getLocalArray()->ptrDev());
	
	// this will allow for printing while exiting right after 
	// hipDeviceReset(); exit(1);
      }
      else if (m_qAlgoName == "BigMem") {
	// AL: this algorithm is similar to the other one but can avoid using 
	//     atomics by storing much more memory
	
	// both algorithms have similar performance problems...
	computeQKernelExponentialDirBigMem<<<blocksPerDir,nThreadsDir>>> 
	  (dStart, dEnd, nbCells,
	   cellFaces->getPtr()->ptrDev(),
	   m_faceCell.ptrDev(),
	   m_nbFacesInCell.ptrDev(),
	   isOutward.getLocalArray()->ptrDev(),
	   m_advanceOrder.ptrDev(),
	   m_weight.ptrDev(),
	   volumes.getLocalArray()->ptrDev(),
	   m_fieldSource.ptrDev(),
	   m_fieldAbsor.ptrDev(),
	   normals.getLocalArray()->ptrDev(),
	   m_dirs.ptrDev(),
	   m_InDir.ptrDev(),
	   m_divqDir.ptrDev(),
	   m_qxDir.ptrDev(),
	   m_qyDir.ptrDev(),
	   m_qzDir.ptrDev());
	
	// this will allow for printing while exiting right after 
	// hipDeviceReset(); exit(1);
      }
      
      /*const CFuint nbKernelCalls = std::ceil((CFreal)nbCells/(CFreal)SHARED_ARRAY_SIZE); 
	
	CFLog(VERBOSE, "RadiativeTransferFVDOMCUDA::loopOverBins() => nbBlocks["
	<< blocksPerDir <<"], nbThreads[" << nThreadsDir 
	<< "], nbKernelCalls = " << nbKernelCalls <<"\n");
	
	CFuint startCellID = 0;
	CFuint endCellID = std::min(nbCells, (CFuint) SHARED_ARRAY_SIZE);
	for (CFuint k = 0; k < nbKernelCalls; ++k) {
	computeQKernelExponentialDir<<<blocksPerDir,nThreadsDir>>> 
	(dStart, dEnd, startCellID, endCellID, nbCells,
	cellFaces->getPtr()->ptrDev(),
	m_faceCell.ptrDev(),
	m_nbFacesInCell.ptrDev(),
	isOutward.getLocalArray()->ptrDev(),
	m_advanceOrder.ptrDev(),
	m_weight.ptrDev(),
	volumes.getLocalArray()->ptrDev(),
	m_fieldSource.ptrDev(),
	m_fieldAbsor.ptrDev(),
	normals.getLocalArray()->ptrDev(),
	m_dirs.ptrDev(),
	divQ.getLocalArray()->ptrDev(),
	qx.getLocalArray()->ptrDev(),
	qy.getLocalArray()->ptrDev(),
	qz.getLocalArray()->ptrDev());
	
	startCellID = endCellID;
	endCellID = std::min(nbCells, endCellID + SHARED_ARRAY_SIZE);
	}*/
    }
    else {
      computeQKernelNoExponentialDir<<<blocksPerDir,nThreadsDir>>> 
	(dStart, dEnd, nbCells,
	 cellFaces->getPtr()->ptrDev(),
	 m_faceCell.ptrDev(),
	 m_nbFacesInCell.ptrDev(),
	 isOutward.getLocalArray()->ptrDev(),
	 m_advanceOrder.ptrDev(),
	 m_weight.ptrDev(),
	 volumes.getLocalArray()->ptrDev(),
	 m_fieldSource.ptrDev(),
	 m_fieldAbSrcV.ptrDev(),
	 m_fieldAbV.ptrDev(),
	 normals.getLocalArray()->ptrDev(),
	 m_dirs.ptrDev(),
	 m_InDir.ptrDev(), 
	 divQ.getLocalArray()->ptrDev(),
	 qx.getLocalArray()->ptrDev(),
	 qy.getLocalArray()->ptrDev(),
	 qz.getLocalArray()->ptrDev());
    }
    
    for (CFuint d = dStart; d < dEnd; ++d) {
      CFLog(INFO, d << " ");
      //(m_useExponentialMethod) ? 
      //computeQExponential(ib,d) : computeQNoExponential(ib,d);
    }
    CFLog(INFO, ")\n");
  }
  
  CFLog(INFO, "RadiativeTransferFVDOMCUDA::loopOverBins() => loop took " << stp.read() << "s \n");
  
  stp.start();
  
  if (m_qAlgoName == "Atomic") {
    socket_divq.getDataHandle().getLocalArray()->get();
    socket_qx.getDataHandle().getLocalArray()->get();
    socket_qy.getDataHandle().getLocalArray()->get();
    socket_qz.getDataHandle().getLocalArray()->get();
  }
  else if (m_qAlgoName == "BigMem") {
    m_qxDir.get(); 
    m_qyDir.get();
    m_qzDir.get();
    m_divqDir.get();
   
    // this is only working on single GPU
    // neds to be fixed for multi-GPU  
    const CFuint dStart = 0; 
    const CFuint dEnd = m_nbDirs; 
    for (CFuint c = 0; c < nbCells; ++c) {
      qx[c] = qy[c] = qz[c] = divQ[c] = 0.;
      const CFuint startc = c*m_nbDirs;
      for (CFuint d = dStart; d < dEnd; ++d) {
	const CFuint startd = startc+d;
	qx[c]   += m_qxDir[startd];
        qy[c]   += m_qyDir[startd];
        qz[c]   += m_qzDir[startd];
	divQ[c] += m_divqDir[startd];
      }
    }
  }
  
  CFLog(INFO, "RadiativeTransferFVDOMCUDA::loopOverBins() => GPU-CPU transfer took " << stp.read() << "s \n");
  
  /* for (CFuint k = 0; k < socket_divq.getDataHandle().size(); ++k) {
     CFLog(INFO, "divQ[" <<k << "] => (" << socket_divq.getDataHandle()[k] << "\n");
     }
     for (CFuint k = 0; k < socket_qx.getDataHandle().size(); ++k) {
     CFLog(INFO, "qx[" <<k << "] => (" << socket_qx.getDataHandle()[k] << "\n");
     }
     for (CFuint k = 0; k < socket_qy.getDataHandle().size(); ++k) {
     CFLog(INFO, "qy[" <<k << "] => (" << socket_qy.getDataHandle()[k] << "\n");
     }
     for (CFuint k = 0; k < socket_qz.getDataHandle().size(); ++k) {
     CFLog(INFO, "qz[" <<k << "] => (" << socket_qz.getDataHandle()[k] << "\n");
     }
     exit(1);*/// 
}
      
//////////////////////////////////////////////////////////////////////////////

    } // namespace RadiativeTransfer

} // namespace COOLFluiD

//////////////////////////////////////////////////////////////////////////////

