#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>

#include "Common/PE.hh"
#include "Common/BadValueException.hh"
#include "Common/CFPrintContainer.hh"
#include "Common/CUDA/CudaDeviceManager.hh"
#include "Common/CUDA/CFVec.hh"
#include "Common/CUDA/CudaTimer.hh"

#include "MathTools/MathConsts.hh"

#include "Environment/ObjectProvider.hh"
#include "Environment/CFEnv.hh"
#include "Environment/FileHandlerOutput.hh"
#include "Environment/FileHandlerInput.hh"
#include "Environment/DirPaths.hh"
#include "Environment/SingleBehaviorFactory.hh"

#include "Framework/PathAppender.hh"
#include "Framework/DataProcessing.hh"
#include "Framework/SubSystemStatus.hh"
#include "Framework/MethodCommandProvider.hh"
#include "Framework/MeshData.hh"
#include "Framework/PhysicalChemicalLibrary.hh"
#include "Framework/SocketBundleSetter.hh"

#include "FiniteVolume/CellCenterFVM.hh"

#include "RadiativeTransfer/RadiativeTransfer.hh"
#include "RadiativeTransfer/Solvers/FiniteVolumeDOM/RadiativeTransferFVDOMCUDA.hh"
#include "RadiativeTransfer/RadiationLibrary/Radiator.hh"
#include "RadiativeTransfer/RadiationLibrary/RadiationPhysicsHandler.hh"

/////////////////////////////////////////////////////////////////////////////

using namespace std;
using namespace COOLFluiD::Framework;
using namespace COOLFluiD::Common;
using namespace COOLFluiD::MathTools;
using namespace COOLFluiD::Numerics::FiniteVolume;

//////////////////////////////////////////////////////////////////////////////

namespace COOLFluiD {

    namespace RadiativeTransfer {

//////////////////////////////////////////////////////////////////////////////

MethodCommandProvider<RadiativeTransferFVDOMCUDA, 
		      DataProcessingData, 
		      RadiativeTransferModule>
radiativeTransferFVDOMCUDAProvider("RadiativeTransferFVDOMCUDA");

//////////////////////////////////////////////////////////////////////////////

__device__ void tableInterpolate
(const CFuint nbBins, const CFuint nbTemp, const CFuint nbPress, 
 const CFreal* Ttable, const CFreal* Ptable, const CFreal* opacities, 
 const CFreal* radSource, CFreal T, CFreal p, CFuint ib, CFreal& val1, CFreal& val2)
{
  //Find the lower bound fo the temperature and the pressure ranges
  //we assume that the temperature and pressure always fall in the bounds.
  //If they don't then the value are still interpolated from the nearest
  //two points in the temperature or pressure list
  CFuint it = nbTemp - 2;
  for (CFuint i = 1; i < (nbTemp - 2); i++){
    if(Ttable[i] > T) { it = i - 1; break;}
  }
  
  CFuint ip = nbPress - 2;
  for (CFuint i = 1; i < (nbPress - 2); i++){
    if(Ptable[i] > p) { ip = i - 1; break;}
  }
  
  //Linear interpolation for the pressure
  
  const CFuint iPiBiT           = it + ib*nbTemp + ip*nbBins*nbTemp;
  const CFuint iPplus1iBiT      = it + ib*nbTemp + (ip + 1)*nbBins*nbTemp;
  const CFuint iPiBiTplus1      = (it + 1) + ib*nbTemp + ip*nbBins*nbTemp;
  const CFuint iPplus1iBiTplus1 = (it + 1) + ib*nbTemp + (ip + 1)*nbBins*nbTemp;
  
  // Linear interpolation for the pressure
  // Interpolation of the opacities
  const CFreal bt1op = (opacities[iPplus1iBiT] - opacities[iPiBiT])*
		    (p - Ptable[ip])/(Ptable[ip + 1] - Ptable[ip]) + opacities[iPiBiT];
  
  const CFreal bt2op = (opacities[iPplus1iBiTplus1] - opacities[iPiBiTplus1])*
		    (p - Ptable[ip])/(Ptable[ip + 1] - Ptable[ip]) + opacities[iPiBiTplus1];
  
  // Interpolation of the source
  const CFreal bt1so = (radSource[iPplus1iBiT] - radSource[iPiBiT])*
		    (p - Ptable[ip])/(Ptable[ip + 1] - Ptable[ip]) + radSource[iPiBiT];
  
  const CFreal bt2so = (radSource[iPplus1iBiTplus1] - radSource[iPiBiTplus1])*
		    (p - Ptable[ip])/(Ptable[ip + 1] - Ptable[ip]) + radSource[iPiBiTplus1];    
  
  // Logarithmic interpolation for the temperature
  // Protect against log(0) and x/0 by switching to linear interpolation if either
  // bt1 or bt2 == 0.  (Note we can't allow log of negative numbers either)
  // Interpolation of the opacities   
  if(bt1op <= 0 || bt2op <= 0){
    val1 = (bt2op - bt1op)*(T - Ttable[it])/(Ttable[it + 1] - Ttable[it]) + bt1op;
//    cout <<"\nOption1 \n";
//    cout <<"T = "<< T <<"\tTi+1 = "<<Ttable[it + 1]<<"\tTi = "<<Ttable[it] <<"\n";
//    cout <<"val1 = " << val1 <<"\tbt2op ="<< bt2op <<"\tbt1op ="<< bt1op <<"\n";
  }
  else {
    val1 = std::exp((T - Ttable[it])/(Ttable[it + 1] - Ttable[it])*std::log(bt2op/bt1op))*bt1op;
//     cout <<"\nOption2 \n";
//     cout <<"T = "<< T <<"\tTi+1 = "<<Ttable[it + 1]<<"\tTi = "<<Ttable[it] <<"\n";
//     cout <<"val1 = " << val1 <<"\tbt2op ="<< bt2op <<"\tbt1op ="<< bt1op <<"\n";
  }
  // Interpolation of the source
  if(bt1so <= 0 || bt2so <= 0){
    val2 = (bt2so - bt1so)*(T - Ttable[it])/(Ttable[it + 1] - Ttable[it]) + bt1so;
//     cout <<"\nOption3 \n";
//     cout <<"T = "<< T <<"\tTi+1 = "<<Ttable[it + 1]<<"\tTi = "<<Ttable[it] <<"\n";
//     cout <<"val1 = " << val2 <<"\tbt2so ="<< bt2so <<"\tbt1so ="<< bt1so <<"\n";
  }
  else {
    val2 = std::exp((T - Ttable[it])/(Ttable[it + 1] - Ttable[it])*std::log(bt2so/bt1so))*bt1so;
//     cout <<"\nOption3 \n";
//     cout <<"T = "<< T <<"\tTi+1 = "<<Ttable[it + 1]<<"\tTi = "<<Ttable[it] <<"\n";
//     cout <<"val2 = " << val2 <<"\tbt2so ="<< bt2so <<"\tbt1so ="<< bt1so <<"\n";
  }
  
  //cf_assert(ib == 0);
}
      
//////////////////////////////////////////////////////////////////////////////

__global__ void getFieldOpacitiesKernel(const bool useExponentialMethod,
					const CFuint TID, 
					const CFuint PID,
					const CFuint nbTemp,
					const CFuint nbPress,
					const CFuint nbBins,
					const CFuint ib,
					const CFuint nbEqs,
					const CFuint nbCells,
					const CFreal* states,
					const CFreal* volumes,
					const CFreal* Ttable,
					const CFreal* Ptable,
					const CFreal* opacities,
					const CFreal* radSource,
					CFreal* fieldSource,
					CFreal* fieldAbsor,
					CFreal* fieldAbSrcV,
					CFreal* fieldAbV)
{    
  // each thread takes care of computing the gradient for one single cell
  const int cellID = threadIdx.x + blockIdx.x*blockDim.x;
  
  if (cellID < nbCells) {
    fieldSource[cellID] = 0.;
    if(useExponentialMethod) {
      fieldAbsor[cellID]  = 0.;
    }
    else {
      fieldAbSrcV[cellID] = 0.;
      fieldAbV[cellID]    = 0.;
    }
    
    //Get the field pressure and T commented because now we impose a temperature profile
    const CFuint sIdx = cellID*nbEqs; 
    const CFreal p = states[sIdx + PID];
    const CFreal T = states[sIdx + TID];
    const CFreal patm = p/101325.; //converting from Pa to atm
    
    CFreal val1 = 0;
    CFreal val2 = 0;
    
    tableInterpolate(nbBins, nbTemp, nbPress, Ttable, Ptable,
		     opacities, radSource, T, patm, ib, val1, val2); 
    
    if(useExponentialMethod){
      if (val1 <= 1e-30 || val2 <= 1e-30 ){
	fieldSource[cellID] = 1e-30;
	fieldAbsor[cellID]  = 1e-30;
      }
      else {
	fieldSource[cellID] = val2/val1;
	fieldAbsor[cellID]  = val1;
      }
    }
    else{
      if (val1 <= 1e-30 || val2 <= 1e-30 ){
	fieldSource[cellID] = 1e-30;
	fieldAbV[cellID]    = 1e-30*volumes[cellID]; // Volumen converted from m^3 into cm^3
      }
      else {
	fieldSource[cellID] = val2/val1;
	fieldAbV[cellID]    = val1*volumes[cellID];
      }      
      fieldAbSrcV[cellID]   = fieldSource[cellID]*fieldAbV[cellID];
    }
  }
}

//////////////////////////////////////////////////////////////////////////////

__global__ void computeQKernel(const CFuint ib, 
			       const CFuint nbCells,
			       const CFuint* cellFaces,
			       const CFint* isOutward,
			       const CFint* advanceOrder,
			       const CFreal* states,
			       const CFreal* volumes,
			       const CFreal* fieldSource,
			       const CFreal* fieldAbsor,
			       const CFreal* fieldAbSrcV,
			       const CFreal* fieldAbV)
  
{    
  // each thread takes care of computing the gradient for one single cell
  const int cellID = threadIdx.x + blockIdx.x*blockDim.x;
  
  // __shared__ typename LIMITER::BASE::template DeviceConfigOptions<NOTYPE> s_dcol[32];
  // typename LIMITER::BASE::template DeviceConfigOptions<NOTYPE>* dcol = &s_dcol[threadIdx.x];
  // dcol->init(gdcol);
  
  // __shared__ typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE> s_dcor[32];
  // typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor = &s_dcor[threadIdx.x];
  // dcor->init(gdcor);
        
  if (cellID < nbCells) {
    
  }
}
      
//////////////////////////////////////////////////////////////////////////////

RadiativeTransferFVDOMCUDA::RadiativeTransferFVDOMCUDA(const std::string& name) :
  RadiativeTransferFVDOM(name)
{
  addConfigOptionsTo(this);
}
      
//////////////////////////////////////////////////////////////////////////////

RadiativeTransferFVDOMCUDA::~RadiativeTransferFVDOMCUDA()
{
}

//////////////////////////////////////////////////////////////////////////////

void RadiativeTransferFVDOMCUDA::defineConfigOptions(Config::OptionList& options)
{  
}
      
//////////////////////////////////////////////////////////////////////////////

void RadiativeTransferFVDOMCUDA::setup()
{
  CFAUTOTRACE;
  
  RadiativeTransferFVDOM::setup();
  
  // store invariant data on GPU
  SafePtr<ConnectivityTable<CFuint> > cellFaces = 
    MeshDataStack::getActive()->getConnectivity("cellFaces");
  cellFaces->getPtr()->put();
  socket_isOutward.getDataHandle().getLocalArray()->put(); 
  socket_volumes.getDataHandle().getLocalArray()->put(); 
  
  m_fieldSource.put();
  m_fieldAbsor.put();
  m_fieldAbSrcV.put();
  m_fieldAbV.put();  
  m_In.put();
  m_II.put();
  m_opacities.put();
  m_radSource.put();
  m_Ttable.put();
  m_Ptable.put();
}
      
//////////////////////////////////////////////////////////////////////////////

void RadiativeTransferFVDOMCUDA::unsetup()
{
  CFAUTOTRACE;
  
  RadiativeTransferFVDOM::unsetup();
}
      
//////////////////////////////////////////////////////////////////////////////

/*void RadiativeTransferFVDOMCUDA::computeQ(const CFuint ib, const CFuint d)
{      
  CFLog(VERBOSE, "RadiativeTransferFVDOMCUDA::computeQ() in (bin, dir) = ("
	<< ib << ", " << d << ") => start\n");
  
  DataHandle<CFreal> volumes = socket_volumes.getDataHandle();
  CellTrsGeoBuilder::GeoData& geoData = m_geoBuilder.getDataGE();
  SafePtr<TopologicalRegionSet> cells = geoData.trs;
  const CFuint nbCells = cells->getLocalNbGeoEnts();
  cf_assert(m_advanceOrder[d].size() == nbCells);
  
  // precompute the dot products for all faces and directions (a part from the sign)
  RealVector dotProdInFace;
  computeDotProdInFace(d, dotProdInFace);
  SafePtr<ConnectivityTable<CFuint> > cellFaces = MeshDataStack::getActive()->getConnectivity("cellFaces");
  DataHandle<CFint> isOutward = socket_isOutward.getDataHandle();
  
  for (CFuint m = 0; m < nbCells; m++) {
    CFreal inDirDotnANeg = 0.;
    CFreal Ic            = 0.;
    
    // allocate the cell entity
    const CFuint iCell = std::abs(m_advanceOrder[d][m]);
    
    // new algorithm (more parallelizable): opacities are computed cell by cell
    // for a given bin
    if (!m_oldAlgo) {getFieldOpacities(ib, iCell);} 
    
    const CFuint nbFaces = cellFaces->nbCols(iCell);
    
    if(m_useExponentialMethod){
      inDirDotnANeg = 0.;
      CFreal dirDotnANeg = 0;
      CFreal Lc      = 0;
      CFreal halfExp = 0;
      
      for (CFuint iFace = 0; iFace < nbFaces; ++iFace) {
	const CFuint faceID = (*cellFaces)(iCell, iFace);
	const CFreal factor = ((CFuint)(isOutward[faceID]) != iCell) ? -1. : 1.;
	const CFreal dirDotNA = dotProdInFace[faceID]*factor;
	
	if(dirDotNA < 0.) {
	  dirDotnANeg += dirDotNA;
	  
	  const bool isBFace = m_mapGeoToTrs->isBGeo(faceID);
	  if (!isBFace){
	    const CFuint neighborCellID = getNeighborCellID(faceID, iCell);
	    inDirDotnANeg += m_In[neighborCellID]*dirDotNA;
	  }
	  else {
	    const CFreal boundarySource = m_fieldSource[iCell];
	    inDirDotnANeg += boundarySource*dirDotNA;
	  }
	}
      } 
      Lc          = volumes[iCell]/(- dirDotnANeg); 
      halfExp     = std::exp(-0.5*Lc*m_fieldAbsor[iCell]);
      m_In[iCell] = (inDirDotnANeg/dirDotnANeg)*halfExp*halfExp + (1. - halfExp*halfExp)*m_fieldSource[iCell];
      Ic          = (inDirDotnANeg/dirDotnANeg)*halfExp + (1. - halfExp)*m_fieldSource[iCell];
    }
    else{
      CFreal dirDotnAPos = 0;
      for (CFuint iFace = 0; iFace < nbFaces; ++iFace) {
	const CFuint faceID = (*cellFaces)(iCell, iFace);
	const CFreal factor = ((CFuint)(isOutward[faceID]) != iCell) ? -1. : 1.;
	const CFreal dirDotNA = dotProdInFace[faceID]*factor;
	
	if (dirDotNA >= 0.){
	  dirDotnAPos += dirDotNA;
	}
	else {
	  const bool isBFace = m_mapGeoToTrs->isBGeo(faceID);
	  if (!isBFace){
	    const CFuint neighborCellID = getNeighborCellID(faceID, iCell);
	    inDirDotnANeg += m_In[neighborCellID]*dirDotNA;
	  }
	  else {
	    const CFreal boundarySource = m_fieldSource[iCell];
	    inDirDotnANeg += boundarySource*dirDotNA;
	  }
	}
      } 
      m_In[iCell] = (m_fieldAbSrcV[iCell] - inDirDotnANeg)/(m_fieldAbV[iCell] + dirDotnAPos);
      Ic = m_In[iCell];
    }
    
    m_q(iCell,XX) += Ic*m_dirs(d,0)*m_weight[d];
    m_q(iCell,YY) += Ic*m_dirs(d,1)*m_weight[d];
    m_q(iCell,ZZ) += Ic*m_dirs(d,2)*m_weight[d];
    
    CFreal inDirDotnA = inDirDotnANeg;
    for (CFuint iFace = 0; iFace < nbFaces; ++iFace) {
      const CFuint faceID = (*cellFaces)(iCell, iFace);
      const CFreal factor = ((CFuint)(isOutward[faceID]) != iCell) ? -1. : 1.;
      const CFreal dirDotNA = dotProdInFace[faceID]*factor;
      if (dirDotNA > 0.) {
	inDirDotnA += m_In[iCell]*dirDotNA;
      }
    }
    
    m_divq[iCell] += inDirDotnA*m_weight[d];
    m_II[iCell]   += Ic*m_weight[d];
  }  
  
  CFLog(VERBOSE, "RadiativeTransferFVDOMCUDA::computeQ() in (bin, dir) = ("
	<< ib << ", " << d << ") => end\n");
}*/
      
//////////////////////////////////////////////////////////////////////////////

 
void RadiativeTransferFVDOMCUDA::loopOverDirs(const CFuint startBin, 
					      const CFuint endBin, 
					      const CFuint startDir,
					      const CFuint endDir)
{
  SafePtr<ConnectivityTable<CFuint> > cellFaces = 
    MeshDataStack::getActive()->getConnectivity("cellFaces");
  DataHandle<CFint> isOutward = socket_isOutward.getDataHandle();
  DataHandle<CFreal> volumes = socket_volumes.getDataHandle();
  DataHandle<State*, GLOBAL> states = socket_states.getDataHandle();
  const CFuint nbCells = states.size();
  const CFuint nbEqs = PhysicalModelStack::getActive()->getNbEq();
  
  states.getGlobalArray()->put();
  
  const CFuint blocksPerGrid = 
    CudaEnv::CudaDeviceManager::getInstance().getBlocksPerGrid(nbCells);
  const CFuint nThreads = CudaEnv::CudaDeviceManager::getInstance().getNThreads();
  
  for (CFuint d = startDir; d < endDir; ++d) {
    CFLog(INFO, "( dir: " << d << " ), ( bin: ");
    const CFuint bStart = (d != startDir) ? 0 : startBin;
    const CFuint bEnd   = (d != m_startEndDir.second) ? m_nbBins : endBin;
    
    m_advanceOrder[d].put();
    
    for(CFuint ib = startBin; ib < endBin; ++ib) {
      CFLog(INFO, "[dir, bin] = [" << d << ", " << ib << "]\n");
      
      // precompute the radiation properties for all cells
      getFieldOpacitiesKernel<<<blocksPerGrid,nThreads>>>
	(m_useExponentialMethod, 
	 m_TID, m_PID, m_nbTemp, m_nbPress, m_nbBins,
	 ib, nbEqs, nbCells, m_Ttable.ptrDev(), m_Ptable.ptrDev(), 
	 states.getGlobalArray()->ptrDev(),
	 volumes.getLocalArray()->ptrDev(),
	 m_opacities.ptrDev(),
	 m_radSource.ptrDev(),
	 m_fieldSource.ptrDev(),
	 m_fieldAbsor.ptrDev(),
	 m_fieldAbSrcV.ptrDev(),
	 m_fieldAbV.ptrDev());  
      
      // compute the radiative heat flux
      computeQKernel<<<blocksPerGrid,nThreads>>> 
	(ib, nbCells,
	 cellFaces->getPtr()->ptrDev(),
	 isOutward.getLocalArray()->ptrDev(),
	 m_advanceOrder[d].ptrDev(),
	 states.getGlobalArray()->ptrDev(),
	 volumes.getLocalArray()->ptrDev(),
	 m_fieldSource.ptrDev(),
	 m_fieldAbsor.ptrDev(),
	 m_fieldAbSrcV.ptrDev(),
	 m_fieldAbV.ptrDev());
    }    
    
    CFLog(INFO, ")\n");
  }
}

//////////////////////////////////////////////////////////////////////////////

    } // namespace RadiativeTransfer

} // namespace COOLFluiD

//////////////////////////////////////////////////////////////////////////////

