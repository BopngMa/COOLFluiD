#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>

#include "Common/PE.hh"
#include "Common/BadValueException.hh"
#include "Common/CFPrintContainer.hh"
#include "Common/CUDA/CudaDeviceManager.hh"
#include "Common/CUDA/CFVec.hh"
#include "Common/CUDA/CudaTimer.hh"

#include "MathTools/MathConsts.hh"

#include "Environment/ObjectProvider.hh"
#include "Environment/CFEnv.hh"
#include "Environment/FileHandlerOutput.hh"
#include "Environment/FileHandlerInput.hh"
#include "Environment/DirPaths.hh"
#include "Environment/SingleBehaviorFactory.hh"

#include "Framework/PathAppender.hh"
#include "Framework/DataProcessing.hh"
#include "Framework/SubSystemStatus.hh"
#include "Framework/MethodCommandProvider.hh"
#include "Framework/MeshData.hh"
#include "Framework/PhysicalChemicalLibrary.hh"
#include "Framework/SocketBundleSetter.hh"

#include "FiniteVolume/CellCenterFVM.hh"

#include "RadiativeTransfer/RadiativeTransfer.hh"
#include "RadiativeTransfer/Solvers/FiniteVolumeDOM/RadiativeTransferFVDOMCUDA.hh"
#include "RadiativeTransfer/RadiationLibrary/Radiator.hh"
#include "RadiativeTransfer/RadiationLibrary/RadiationPhysicsHandler.hh"

/////////////////////////////////////////////////////////////////////////////

using namespace std;
using namespace COOLFluiD::Framework;
using namespace COOLFluiD::Common;
using namespace COOLFluiD::MathTools;
using namespace COOLFluiD::Numerics::FiniteVolume;

//////////////////////////////////////////////////////////////////////////////

namespace COOLFluiD {

    namespace RadiativeTransfer {

//////////////////////////////////////////////////////////////////////////////

MethodCommandProvider<RadiativeTransferFVDOMCUDA, 
		      DataProcessingData, 
		      RadiativeTransferModule>
radiativeTransferFVDOMCUDAProvider("RadiativeTransferFVDOMCUDA");

//////////////////////////////////////////////////////////////////////////////

__global__ void getFieldOpacitiesKernel(const bool useExponentialMethod,
					const CFuint TID, 
					const CFuint PID,
					const CFuint nbTemp,
					const CFuint nbPress,
					const CFuint nbBins,
					const CFuint ib,
					const CFuint nbEqs,
					const CFuint nbCells,
					const CFreal* states,
					const CFreal* volumes,
					const CFreal* Ttable,
					const CFreal* Ptable,
					const CFreal* opacities,
					const CFreal* radSource,
					CFreal* fieldSource,
					CFreal* fieldAbsor,
					CFreal* fieldAbSrcV,
					CFreal* fieldAbV)
{    
  // each thread takes care of computing the gradient for one single cell
  const int cellID = threadIdx.x + blockIdx.x*blockDim.x;
  
  if (cellID < nbCells) {
    fieldSource[cellID] = 0.;
    if(useExponentialMethod) {
      fieldAbsor[cellID]  = 0.;
    }
    else {
      fieldAbSrcV[cellID] = 0.;
      fieldAbV[cellID]    = 0.;
    }
    
    //Get the field pressure and T commented because now we impose a temperature profile
    const CFuint sIdx = cellID*nbEqs; 
    const CFreal p = states[sIdx + PID];
    const CFreal T = states[sIdx + TID];
    const CFreal patm = p/101325.; //converting from Pa to atm
    
    CFreal val1 = 0;
    CFreal val2 = 0;
    
    RadiativeTransferFVDOM::Interpolator interp;
    interp.tableInterpolate(nbBins, nbTemp, nbPress, Ttable, Ptable,
			    opacities, radSource, T, patm, ib, val1, val2); 
    
    if(useExponentialMethod){
      if (val1 <= 1e-30 || val2 <= 1e-30 ){
	fieldSource[cellID] = 1e-30;
	fieldAbsor[cellID]  = 1e-30;
      }
      else {
	fieldSource[cellID] = val2/val1;
	fieldAbsor[cellID]  = val1;
      }
    }
    else{
      if (val1 <= 1e-30 || val2 <= 1e-30 ){
	fieldSource[cellID] = 1e-30;
	fieldAbV[cellID]    = 1e-30*volumes[cellID]; // Volumen converted from m^3 into cm^3
      }
      else {
	fieldSource[cellID] = val2/val1;
	fieldAbV[cellID]    = val1*volumes[cellID];
      }      
      fieldAbSrcV[cellID]   = fieldSource[cellID]*fieldAbV[cellID];
    }
  }
}

//////////////////////////////////////////////////////////////////////////////

__global__ void computeQKernelExponential(const CFuint nbCells,
					  const CFreal weightIn,
					  const CFuint* cellFaces,
					  const CFint* isOutward,
					  const CFint* advanceOrder,
					  const CFreal* volumes,
					  const CFreal* fieldSource,
					  const CFreal* fieldAbsor,
					  const CFreal* dotProdInFace,
					  CFreal* In,
					  CFreal* II,
					  CFreal* divQ)
{    
  // each thread takes care of computing the gradient for one single cell
  const int cellID = threadIdx.x + blockIdx.x*blockDim.x;
  
  __shared__ CFreal weight;
  weight = weightIn;
      
  if (cellID < nbCells) {
    // allocate the cell entity
    const CFuint iCell   = abs(advanceOrder[cellID]);
    CFreal Ic            = 0.;
    CFreal inDirDotnANeg = 0.;
    CFreal dirDotnANeg   = 0;
    CFreal Lc            = 0;
    CFreal halfExp       = 0;
    
    const CFuint nbFaces = 5; //// cellFaces->nbCols(iCell);  /////
    for (CFuint iFace = 0; iFace < nbFaces; ++iFace) { 
      const CFuint faceID = cellFaces[iFace*nbCells + iCell];
      const CFreal factor = ((CFuint)(isOutward[faceID]) != iCell) ? -1. : 1.;
      const CFreal dirDotNA = dotProdInFace[faceID]*factor;
      
      if(dirDotNA < 0.) {
	dirDotnANeg += dirDotNA;
        // const CFint fcellID = faceNeighborID[faceID*2]; 
        // const CFint neighborCellID = (fcellID == iCell) ? faceNeighborID[faceID*2+1] : fcellID;
	// const CFreal source = (neighborCellID >=0) ? In[neighborCellID] : fieldSource[iCell];
        // inDirDotnANeg +=source*dirDotNA;
	
	
	/*const bool isBFace = mapGeoToTrs->isBGeo(faceID); /////
	  if (!isBFace){
	  const CFuint neighborCellID = getNeighborCellID(faceID, iCell); /////
	  inDirDotnANeg += In[neighborCellID]*dirDotNA;
	  }
	  else {
	  const CFreal boundarySource = fieldSource[iCell];
	  inDirDotnANeg += boundarySource*dirDotNA;
	  }*/
      }
    } 
    Lc          = volumes[iCell]/(- dirDotnANeg); 
    halfExp     = std::exp(-0.5*Lc*fieldAbsor[iCell]);
    In[iCell] = (inDirDotnANeg/dirDotnANeg)*halfExp*halfExp + (1. - halfExp*halfExp)*fieldSource[iCell];
    Ic          = (inDirDotnANeg/dirDotnANeg)*halfExp + (1. - halfExp)*fieldSource[iCell];
    
    // m_q(iCell,XX) += Ic*m_dirs(d,0)*weight; /////
    // m_q(iCell,YY) += Ic*m_dirs(d,1)*weight; /////
    // m_q(iCell,ZZ) += Ic*m_dirs(d,2)*weight; /////
    
    CFreal inDirDotnA = inDirDotnANeg;
    for (CFuint iFace = 0; iFace < nbFaces; ++iFace) {
      const CFuint faceID = cellFaces[iFace*nbCells + iCell];
      const CFreal factor = ((CFuint)(isOutward[faceID]) != iCell) ? -1. : 1.;
      const CFreal dirDotNA = dotProdInFace[faceID]*factor;
      if (dirDotNA > 0.) {
	inDirDotnA += In[iCell]*dirDotNA;
      }
    }
    
    divQ[iCell] += inDirDotnA*weight;
    II[iCell]   += Ic*weight;
  }  
}
      
//////////////////////////////////////////////////////////////////////////////

__global__ void computeQKernelNoExponential(const CFuint nbCells,
					    const CFreal weight,
					    const CFuint* cellFaces,
					    const CFint* isOutward,
					    const CFint* advanceOrder,
					    const CFreal* volumes,
					    const CFreal* fieldSource,
					    const CFreal* fieldAbSrcV,
					    const CFreal* fieldAbV,
					    const CFreal* dotProdInFace,
					    CFreal* In,
					    CFreal* II,
					    CFreal* divQ)
{    
  // each thread takes care of computing the gradient for one single cell
  const int cellID = threadIdx.x + blockIdx.x*blockDim.x;
  
  // __shared__ typename LIMITER::BASE::template DeviceConfigOptions<NOTYPE> s_dcol[32];
  // typename LIMITER::BASE::template DeviceConfigOptions<NOTYPE>* dcol = &s_dcol[threadIdx.x];
  // dcol->init(gdcol);
  
  // __shared__ typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE> s_dcor[32];
  // typename POLYREC::BASE::template DeviceConfigOptions<NOTYPE>* dcor = &s_dcor[threadIdx.x];
  // dcor->init(gdcor);
        
  if (cellID < nbCells) {
    /*  CFreal inDirDotnANeg = 0.;
    CFreal Ic            = 0.;
    
    // allocate the cell entity
    const CFuint iCell = std::abs(m_advanceOrder[cellID]);
      const CFuint nbFaces = cellFaces->nbCols(iCell);
    
    
    
      CFreal dirDotnAPos = 0;
      for (CFuint iFace = 0; iFace < nbFaces; ++iFace) {
	const CFuint faceID = (*cellFaces)(iCell, iFace);
	const CFreal factor = ((CFuint)(isOutward[faceID]) != iCell) ? -1. : 1.;
	const CFreal dirDotNA = dotProdInFace[faceID]*factor;
	
	if (dirDotNA >= 0.){
	  dirDotnAPos += dirDotNA;
	}
	else {
	  const bool isBFace = m_mapGeoToTrs->isBGeo(faceID);
	  if (!isBFace){
	    const CFuint neighborCellID = getNeighborCellID(faceID, iCell);
	    inDirDotnANeg += m_In[neighborCellID]*dirDotNA;
	  }
	  else {
	    const CFreal boundarySource = m_fieldSource[iCell];
	    inDirDotnANeg += boundarySource*dirDotNA;
	  }
	}
      } 
      m_In[iCell] = (m_fieldAbSrcV[iCell] - inDirDotnANeg)/(m_fieldAbV[iCell] + dirDotnAPos);
      Ic = m_In[iCell];
    
    
    m_q(iCell,XX) += Ic*m_dirs(d,0)*m_weight[d];
    m_q(iCell,YY) += Ic*m_dirs(d,1)*m_weight[d];
    m_q(iCell,ZZ) += Ic*m_dirs(d,2)*m_weight[d];
    
    CFreal inDirDotnA = inDirDotnANeg;
    for (CFuint iFace = 0; iFace < nbFaces; ++iFace) {
      const CFuint faceID = (*cellFaces)(iCell, iFace);
      const CFreal factor = ((CFuint)(isOutward[faceID]) != iCell) ? -1. : 1.;
      const CFreal dirDotNA = dotProdInFace[faceID]*factor;
      if (dirDotNA > 0.) {
	inDirDotnA += m_In[iCell]*dirDotNA;
      }
    }
    
    divQ[iCell] += inDirDotnA*m_weight[d];
    m_II[iCell]   += Ic*m_weight[d];*/
  }  
}
      
//////////////////////////////////////////////////////////////////////////////

RadiativeTransferFVDOMCUDA::RadiativeTransferFVDOMCUDA(const std::string& name) :
  RadiativeTransferFVDOM(name)
{
  addConfigOptionsTo(this);
}
      
//////////////////////////////////////////////////////////////////////////////

RadiativeTransferFVDOMCUDA::~RadiativeTransferFVDOMCUDA()
{
}

//////////////////////////////////////////////////////////////////////////////

void RadiativeTransferFVDOMCUDA::defineConfigOptions(Config::OptionList& options)
{  
}
      
//////////////////////////////////////////////////////////////////////////////

void RadiativeTransferFVDOMCUDA::setup()
{
  CFAUTOTRACE;
  
  RadiativeTransferFVDOM::setup();
  
  // store invariant data on GPU
  SafePtr<ConnectivityTable<CFuint> > cellFaces = 
    MeshDataStack::getActive()->getConnectivity("cellFaces");
  cellFaces->getPtr()->put();
  socket_isOutward.getDataHandle().getLocalArray()->put(); 
  socket_volumes.getDataHandle().getLocalArray()->put(); 
  socket_divq.getDataHandle().getLocalArray()->put(); 
  
  m_fieldSource.put();
  m_fieldAbsor.put();
  m_fieldAbSrcV.put();
  m_fieldAbV.put();  
  m_In.put();
  m_II.put();
  m_opacities.put();
  m_radSource.put();
  m_Ttable.put();
  m_Ptable.put();
}
      
//////////////////////////////////////////////////////////////////////////////

void RadiativeTransferFVDOMCUDA::unsetup()
{
  CFAUTOTRACE;
  
  RadiativeTransferFVDOM::unsetup();
}
      
//////////////////////////////////////////////////////////////////////////////
 
void RadiativeTransferFVDOMCUDA::loopOverDirs(const CFuint startBin, 
					      const CFuint endBin, 
					      const CFuint startDir,
					      const CFuint endDir)
{
  SafePtr<ConnectivityTable<CFuint> > cellFaces = 
    MeshDataStack::getActive()->getConnectivity("cellFaces");
  DataHandle<CFint> isOutward = socket_isOutward.getDataHandle();
  DataHandle<CFreal> volumes = socket_volumes.getDataHandle();
  DataHandle<CFreal> divQ = socket_divq.getDataHandle();
  DataHandle<State*, GLOBAL> states = socket_states.getDataHandle();
  const CFuint nbCells = states.size();
  const CFuint nbEqs = PhysicalModelStack::getActive()->getNbEq();
  
  states.getGlobalArray()->put();
  
  const CFuint blocksPerGrid = 
    CudaEnv::CudaDeviceManager::getInstance().getBlocksPerGrid(nbCells);
  const CFuint nThreads = CudaEnv::CudaDeviceManager::getInstance().getNThreads();
  
  for (CFuint d = startDir; d < endDir; ++d) {
    CFLog(INFO, "( dir: " << d << " ), ( bin: ");
    const CFuint bStart = (d != startDir) ? 0 : startBin;
    const CFuint bEnd   = (d != m_startEndDir.second) ? m_nbBins : endBin;
    
    m_advanceOrder[d].put();
    
    // precompute dot products for all faces and directions (a part from the sign)
    computeDotProdInFace(d, m_dotProdInFace);
    m_dotProdInFace.put();
    
    for (CFuint ib = startBin; ib < endBin; ++ib) {
      CFLog(INFO, "[dir, bin] = [" << d << ", " << ib << "]\n");
      
      // precompute the radiation properties for all cells
      getFieldOpacitiesKernel<<<blocksPerGrid,nThreads>>>
	(m_useExponentialMethod, 
	 m_TID, m_PID, m_nbTemp, m_nbPress, m_nbBins,
	 ib, nbEqs, nbCells, m_Ttable.ptrDev(), m_Ptable.ptrDev(), 
	 states.getGlobalArray()->ptrDev(),
	 volumes.getLocalArray()->ptrDev(),
	 m_opacities.ptrDev(),
	 m_radSource.ptrDev(),
	 m_fieldSource.ptrDev(),
	 m_fieldAbsor.ptrDev(),
	 m_fieldAbSrcV.ptrDev(),
	 m_fieldAbV.ptrDev());  
      
      // m_fieldSource.get();
      //  m_fieldAbsor.get();
      // m_fieldAbSrcV.get();
      // m_fieldAbV.get();
      
      // RadiativeTransferFVDOM::computeQ(ib,d);
      
      // compute the radiative heat flux
      if (m_useExponentialMethod) {
	computeQKernelExponential<<<blocksPerGrid,nThreads>>> 
	  (nbCells, m_weight[d],
	   cellFaces->getPtr()->ptrDev(),
	   isOutward.getLocalArray()->ptrDev(),
	   m_advanceOrder[d].ptrDev(),
	   volumes.getLocalArray()->ptrDev(),
	   m_fieldSource.ptrDev(),
	   m_fieldAbsor.ptrDev(),
	   m_dotProdInFace.ptrDev(),
	   m_In.ptrDev(), m_II.ptrDev(), divQ.getLocalArray()->ptrDev());
      }
      else {
	computeQKernelNoExponential<<<blocksPerGrid,nThreads>>> 
	  (nbCells, m_weight[d],
	   cellFaces->getPtr()->ptrDev(),
	   isOutward.getLocalArray()->ptrDev(),
	   m_advanceOrder[d].ptrDev(),
	   volumes.getLocalArray()->ptrDev(),
	   m_fieldSource.ptrDev(),
	   m_fieldAbSrcV.ptrDev(),
	   m_fieldAbV.ptrDev(),
	   m_dotProdInFace.ptrDev(),
	   m_In.ptrDev(), m_II.ptrDev(), divQ.getLocalArray()->ptrDev());
	
      }
      CFLog(INFO, ")\n");
    }
  }
  socket_divq.getDataHandle().getLocalArray()->get(); 
}

//////////////////////////////////////////////////////////////////////////////

    } // namespace RadiativeTransfer

} // namespace COOLFluiD

//////////////////////////////////////////////////////////////////////////////

